#include "hip/hip_runtime.h"
/* Implement SMHA - this one works!!!! */
/* 9/8/2017 making modifications for linux version */


#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <queue>
#include <string>
#include <iterator>
#include <fstream>
#include <sstream>
#include <algorithm>

#define _USE_MATH_DEFINES
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <limits>
#include <vector>
#include <set>
#include <algorithm>
#include <map>

#include "math.h"
#include "hip/hip_math_constants.h"
#include "smha_headers.hpp"

//////////////////////////////////////////////////////////////////////////////////////////////////////////

using namespace std;

char *behavior_array[DIR] = { "rendezvous", "flocking", "flock_east", "flock_north", "flock_west", "flock_south", "antirendezvous" };

/* Error Checking..... */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////


/* DEVICE FUNCTIONS (BEHAVIOR and KERNEL) */

__device__
float d_wrapToPi(float input)
{
	float result = fmodf(input + HIP_PI_F, 2 * HIP_PI_F);
	result += (result < 0) * 2 * HIP_PI_F;
	result -= HIP_PI_F;
	return result;
}


__device__
void d_robot_model_i(node* current, int i, float u_v, float u_w, float dt)
{
	int N = current->N;
	float max_linear_velocity = 2;
	float max_angular_velocity = HIP_PI_F / 8;
	u_v = fmaxf(-max_linear_velocity, fminf(max_linear_velocity, u_v));
	u_w = fmaxf(-max_angular_velocity, fminf(max_angular_velocity, u_w));
	current->robot_pos[i][0] = current->robot_pos[i][0] + u_v*cosf(current->robot_pos[i][2])*dt;
	current->robot_pos[i][1] = current->robot_pos[i][1] + u_v*sinf(current->robot_pos[i][2])*dt;
	current->robot_pos[i][2] = d_wrapToPi(current->robot_pos[i][2] + u_w*dt);
	return;
}


__device__
void d_antirendezvous(node* future, node *current, int i, float dt)
{
	//printf("Inside rendezvous\n");
	float connectivity_radius = 20;
	float gain_v = 1;
	float gain_w = 1;
	int N = current->N;
	float v[2], dv[2], b[2];
	float w, n;
	float position_i[2], position_j[2], d[2];
	float heading_i, heading_j;
	float dtheta;
	float u_v, u_w;
	position_i[0] = current->robot_pos[i][0];
	position_i[1] = current->robot_pos[i][1];
	heading_i = current->robot_pos[i][2];
	v[0] = 0;
	v[1] = 0;
	w = 0;
	n = 0;
	for (int j = 0; j<N; j++)
	{
		if (i == j) continue;
		position_j[0] = current->robot_pos[j][0];
		position_j[1] = current->robot_pos[j][1];
		heading_j = current->robot_pos[j][2];
		d[0] = position_j[0] - position_i[0];
		d[1] = position_j[1] - position_i[1];
		float norm_d = sqrtf(powf(d[0], 2) + powf(d[1], 2));
		if (norm_d < connectivity_radius)
		{
			dv[0] = -d[0];
			dv[1] = -d[1];
			n = n + 1;
		}
		else
		{
			dv[0] = 0;
			dv[1] = 0;
		}
		v[0] = v[0] + dv[0];
		v[1] = v[1] + dv[1];
	}
	v[0] = v[0] / (n + 1);
	v[1] = v[1] / (n + 1);
	float tmp = atan2f(v[1], v[0]);
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	dtheta = tmp - heading_i;
	w = atan2f(sinf(dtheta), cosf(dtheta));
	w += 2 * HIP_PI_F * (w == -HIP_PI_F);
	b[0] = cosf(heading_i);
	b[1] = sinf(heading_i);
	u_v = gain_v * (v[0] * b[0] + v[1] * b[1]);
	u_w = gain_w * w;
	d_robot_model_i(future, i, u_v, u_w, dt);
	return;
}


__device__
void d_rendezvous(node* future, node *current, int i, float dt)
{
	//printf("Inside rendezvous\n");
	float connectivity_radius = 20;
	float gain_v = 1;
	float gain_w = 1;
	int N = current->N;
	float v[2], dv[2], b[2];
	float w, n;
	float position_i[2], position_j[2], d[2];
	float heading_i, heading_j;
	float dtheta;
	float u_v, u_w;
	position_i[0] = current->robot_pos[i][0];
	position_i[1] = current->robot_pos[i][1];
	heading_i = current->robot_pos[i][2];
	v[0] = 0;
	v[1] = 0;
	w = 0;
	n = 0;
	for (int j = 0; j<N; j++)
	{
		if (i == j) continue;
		position_j[0] = current->robot_pos[j][0];
		position_j[1] = current->robot_pos[j][1];
		heading_j = current->robot_pos[j][2];
		d[0] = position_j[0] - position_i[0];
		d[1] = position_j[1] - position_i[1];
		float norm_d = sqrtf(powf(d[0], 2) + powf(d[1], 2));
		if (norm_d < connectivity_radius)
		{
			dv[0] = d[0];
			dv[1] = d[1];
			n = n + 1;
		}
		else
		{
			dv[0] = 0;
			dv[1] = 0;
		}
		v[0] = v[0] + dv[0];
		v[1] = v[1] + dv[1];
	}
	v[0] = v[0] / (n + 1);
	v[1] = v[1] / (n + 1);
	float tmp = atan2f(v[1], v[0]);
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	dtheta = tmp - heading_i;
	w = atan2f(sinf(dtheta), cosf(dtheta));
	w += 2 * HIP_PI_F * (w == -HIP_PI_F);
	b[0] = cosf(heading_i);
	b[1] = sinf(heading_i);
	u_v = gain_v * (v[0] * b[0] + v[1] * b[1]);
	u_w = gain_w * w;
	d_robot_model_i(future, i, u_v, u_w, dt);
	return;
}

__device__
void d_flock_biased(node* future, node *current, int i, float* bias, float dt)
{
	float min_linear_velocity = 1;
	float repulsion_radius = 5;
	float alignment_radius = 10;
	float attraction_radius = 20;
	float gain_v = 1;
	float gain_w = 1;
	float position_i[2], position_j[2];
	float heading_i, heading_j;
	float v[2], d[2], dv[2], b[2];
	float w, n, dtheta;
	float norm_d;
	float u_v, u_w;
	int N = current->N;
	position_i[0] = current->robot_pos[i][0];
	position_i[1] = current->robot_pos[i][1];
	heading_i = current->robot_pos[i][2];
	v[0] = 0;
	v[1] = 0;
	w = 0;
	n = 0;

	for (int j = 0; j < N; j++)
	{
		if (i == j) continue;
		position_j[0] = current->robot_pos[j][0];
		position_j[1] = current->robot_pos[j][1];
		heading_j = current->robot_pos[j][2];
		n = n + 1;
		d[0] = position_j[0] - position_i[0];
		d[1] = position_j[1] - position_i[1];
		norm_d = sqrtf(powf(d[0], 2) + powf(d[1], 2));
		dtheta = 0;
		if (norm_d < repulsion_radius)
		{
			dv[0] = -d[0] / powf(norm_d, 2);
			dv[1] = -d[1] / powf(norm_d, 2);
			float tmp = atan2f(dv[1], dv[0]);
			tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
			dtheta = tmp - heading_i;
		}
		else if (norm_d < alignment_radius)
		{
			dv[0] = 0;
			dv[1] = 0;
			dtheta = heading_j - heading_i;
		}
		else if (norm_d < attraction_radius)
		{
			dv[0] = d[0];
			dv[1] = d[1];
			float tmp = atan2f(dv[1], dv[0]);
			tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
			dtheta = tmp - heading_i;
		}
		else
		{
			dv[0] = 0;
			dv[1] = 0;
			dtheta = 0;
			n = n - 1;
		}
		v[0] += dv[0];
		v[1] += dv[1];
		float tmp = atan2f(sinf(dtheta), cosf(dtheta));
		tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
		w = w + tmp;
	}
	v[0] = v[0] / (n + 1) + bias[0]; //incorporated Sasanka's line 51 and 54 at once
	v[1] = v[1] / (n + 1) + bias[1];
	w = d_wrapToPi(w / (n + 1));

	float tmp = atan2f(bias[1], bias[0]);
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	dtheta = tmp - heading_i;

	tmp = atan2f(sinf(dtheta), cosf(dtheta));
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	w = w + tmp;

	b[0] = cosf(heading_i);
	b[1] = sinf(heading_i);
	u_v = gain_v * (v[0] * b[0] + v[1] * b[1]);
	u_w = gain_w * w;
	u_v = fmaxf(min_linear_velocity, u_v);

	d_robot_model_i(future, i, u_v, u_w, dt);
	return;
}


__device__
void d_flocking(node* future, node* current, int i, float dt)
{
	float bias[2] = { 0, 0 };
	d_flock_biased(future, current, i, bias, dt);
	return;
}


__device__
void d_flock_east(node* future, node* current, int i, float dt)
{
	float direction[2] = { 1, 0 };
	float gain = 1.0;
	float bias[2];
	bias[0] = gain * direction[0];
	bias[1] = gain * direction[1];
	d_flock_biased(future, current, i, bias, dt);
	return;
}


__device__
void d_flock_north(node* future, node* current, int i, float dt)
{
	float direction[2] = { 0, 1 };
	float gain = 1.0;
	float bias[2];
	bias[0] = gain * direction[0];
	bias[1] = gain * direction[1];
	d_flock_biased(future, current, i, bias, dt);
	return;
}

__device__
void d_flock_west(node* future, node* current, int i, float dt)
{
	float direction[2] = { -1, 0 };
	float gain = 1.0;
	float bias[2];
	bias[0] = gain * direction[0];
	bias[1] = gain * direction[1];
	d_flock_biased(future, current, i, bias, dt);
	return;
}

__device__
void d_flock_south(node* future, node* current, int i, float dt)
{
	float direction[2] = { 0, -1 };
	float gain = 1.0;
	float bias[2];
	bias[0] = gain * direction[0];
	bias[1] = gain * direction[1];
	d_flock_biased(future, current, i, bias, dt);
	return;
}

__device__
void d_move_stop(node* future, node* current, int i, float dt)
{
	return;
}


typedef void(*op_func) (node*, node*, int, float);
__device__ op_func func[DIR] = { d_rendezvous, d_flocking, d_flock_east, d_flock_north, d_flock_west, d_flock_south, d_antirendezvous };
__device__ char *d_behavior_array[DIR] = { "rendezvous", "flocking", "flock_east", "flock_north", "flock_west", "flock_south", "antirendezvous" };


/* returns@ 0 is target is not reached; else 1 */
__device__
int d_target_reached(node curr_node, PARAM* param)
{
	int N = curr_node.N;
	float robot_radius = param->robot_radius;
	float target_radius = param->target_radius;
	/* L2norm(robot_pos(i) - target_center) */
	float robot_x, robot_y;
	float target_x = param->target_center[0];
	float target_y = param->target_center[1];
	float distance;
	for (int i = 0; i < N; i++)
	{
		robot_x = curr_node.robot_pos[i][0];
		robot_y = curr_node.robot_pos[i][1];
		distance = sqrt(powf(robot_x - target_x, 2) + powf(robot_y - target_y, 2));
		if (distance >= target_radius - robot_radius) return 0;
	}
	return 1;
}


/* returns@ 0 is not valid; else 1 */
__device__
int d_valid_poses(node curr_node, PARAM* param)
{
	int i, j;
	int N = curr_node.N;
	int M = param->M;
	float robot_x, robot_y;
	float robot_radius = param->robot_radius;
	float mapsize = param->mapsize;
	float safety_bounds = param->robot_radius;
	/* check if position is off the map */
	for (i = 0; i< N; i++)
	{
		robot_x = curr_node.robot_pos[i][0];
		robot_y = curr_node.robot_pos[i][1];
		if (robot_x > mapsize || robot_y > mapsize ||
			robot_x < -mapsize || robot_y < -mapsize) {
			return 0;
		}
	}
	/* check if collided with obstacle */
	float obs_x, obs_y, obs_r;
	float distance;
	for (i = 0; i < N; i++)
	{
		robot_x = curr_node.robot_pos[i][0];
		robot_y = curr_node.robot_pos[i][1];
		for (j = 0; j < M; j++)
		{
			obs_x = param->obstacle_pos[j][0];
			obs_y = param->obstacle_pos[j][1];
			obs_r = param->obstacle_pos[j][2];
			distance = sqrt(powf(robot_x - obs_x, 2) + powf(robot_y - obs_y, 2));
			if (distance < obs_r + robot_radius + safety_bounds) {
				return 0;
			}
		}
	}
	return 1;
}




__device__
float d_calculate_G(node d_expanded, node d_open, PARAM* d_param, int mode)
{


	float final_G = d_open.G;
	float poses_delta_summed[2];
	poses_delta_summed[0] = 0;
	poses_delta_summed[1] = 0;
	float N = d_open.N;
	for (int i = 0; i < N; i++)
	{
		final_G += sqrtf(powf(d_expanded.robot_pos[i][0] - d_open.robot_pos[i][0], 2) +
			powf(d_expanded.robot_pos[i][1] - d_open.robot_pos[i][1], 2));
	}
	return final_G;
}



/* calculate the heuristic between curr node and the destination */
/* cost_to_go is distance left: cost_to_go = sum(max(pdist2(poses_next(:,1:2),target_center') - (target_radius-robot_radius),0)); */
__device__
float d_calculate_H(node d_expanded, PARAM* d_param, int mode)
{
	float priority = 0;
	float priority_i;
	int i;
	float target_radius = d_param->target_radius;
	float target_center[2];
	target_center[0] = d_param->target_center[0];
	target_center[1] = d_param->target_center[1];
	float robot_radius = d_param->robot_radius;
	int N = d_param->N;
	for (int i = 0; i < N; i++)
	{
		priority_i = sqrt(powf(d_expanded.robot_pos[i][0] - target_center[0], 2)
			+ powf(d_expanded.robot_pos[i][1] - target_center[1], 2));
		priority_i -= target_radius - robot_radius;
		priority_i = fmaxf(priority_i, 0);
		priority += priority_i;
	}
	if (mode == 0) return priority;
	else if (mode == 1) {
		float robot_cx = 0;
		float robot_cy = 0;
		for (int i = 0; i < d_param->N; i++)
		{
			robot_cx += d_expanded.robot_pos[i][0];
			robot_cy += d_expanded.robot_pos[i][1];
		}
		robot_cx /= d_param->N;
		robot_cy /= d_param->N;
		float dist_to_ob = sqrt(powf(robot_cx - d_param->obstacle_pos[0][0], 2) + powf(robot_cy - d_param->obstacle_pos[0][1], 2)) - d_param->obstacle_pos[0][2];;
		for (int j = 0; j < d_param->M; j++)
		{
			dist_to_ob = fminf(dist_to_ob, sqrt(powf(robot_cx - d_param->obstacle_pos[j][0], 2) + powf(robot_cy - d_param->obstacle_pos[j][1], 2)) - d_param->obstacle_pos[0][2]);
		}
		return priority + d_param->N *  powf(3, -(dist_to_ob - 5));
	}

}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////


/* param @ d_expanded - result of expanded nodes
param @ d_open - input of nodes to expand
param @ d_param - parameters about the swarm
param @ dir - Number of directions (7 for now)
param @ iteration - used for backtracing */
__global__
void k_expandStates(node* d_expanded, node* d_open, PARAM* d_param, int dir, int iteration, int real_copies, int queue_i)
{
	int dir_index = threadIdx.x;
	int robot_index = threadIdx.y;
	int node_index = blockIdx.x;
	int index = node_index * dir + dir_index;

	/* Check thread boundary */
	if (dir_index >= dir)  return;
	if (robot_index >= d_param->N) return;
	if (node_index >= real_copies) return;

	/* Expand the nodes */
	float dt = d_param->dt;
	float ti = d_param->ti;
	float dT = d_param->dT;
	float tf = d_param->tf;
 
  if(d_open[node_index].sequence_numel >= d_param->time_array_count + 1) {
    d_expanded[index].isEmpty = 1;
    return;
  }

	int steps = (int)dT / dt;
	int currSequenceIndex = d_open[node_index].sequence_numel;

	if(currSequenceIndex == 0) steps = 
		(int) d_param->time_array[currSequenceIndex] / dt;
  else if(currSequenceIndex == d_param->time_array_count) steps = (int) (tf - d_param->time_array[d_param->time_array_count-1]) / dt;
	else steps = (int) (d_param->time_array[currSequenceIndex] - d_param->time_array[currSequenceIndex - 1]) / dt;
	/* Only write to global memory once */
	if (robot_index == 0) d_expanded[index] = d_open[node_index];

	__syncthreads();

	for (int i = 0; i < steps; i++)
	{
		node d_expanded_old = d_expanded[index];
		func[dir_index](&d_expanded[index], &d_expanded_old, robot_index, dt);
		__syncthreads();
		/* Check swarm map boundary and obstacle collision to separate valid vs invalid */
		if (d_valid_poses(d_expanded[index], d_param) == 0) {
			d_expanded[index].isEmpty = 1;
			return;
		}
		d_expanded[index].G = d_calculate_G(d_expanded[index], d_expanded_old, d_param, queue_i);
	}

	if (robot_index == 0)
	{
		/* Only write to global memory once */
		d_expanded[index].behaviorIndices[currSequenceIndex] = dir_index;
		d_expanded[index].behaviorIdx = d_expanded[index].behaviorIdx * 10 + dir_index;

		/* Check swarm if reached destination */
		d_expanded[index].reached_destination = d_target_reached(d_expanded[index], d_param);

		/* cost_estimate is total cost: cost_estimate = cost_next + H*cost_to_go */
		d_expanded[index].F = d_expanded[index].G + (d_param->H) * d_calculate_H(d_expanded[index], d_param, queue_i);
		d_expanded[index].sequence_numel += 1;
		//printf("INSIDE KERNEL>>> [%d].F = %f, .behaviorIdx = %llu\n", 
    //        index, d_expanded[index].F, d_expanded[index].behaviorIdx);
	}
	return;
}


/////////////////////////////////////////////////////////////////////////////////////////////////////

/* HOST FUNCTIONS */

/* calculate the heuristic between curr node and the destination */
float h_calculate_H1(float robot_pos[ROBOT_MAX][3], PARAM* param, int N)
{
	float priority = 0;
	for (int i = 0; i < N; i++)
	{
		priority += sqrt(pow(robot_pos[i][0] - param->target_center[0], 2)
			+ pow(robot_pos[i][1] - param->target_center[1], 2));
		priority -= param->target_radius - param->robot_radius;
	}

	return priority;
}



////////////////////////////////////////////////////////////////////////////////////////////

/* this function serves to convert position to grid index */
int convert_position(float mapsize, float robot_pos_x, float robot_pos_y)
{
	int col_offset = (int)(robot_pos_x + mapsize);
	int row_offset = (int)(mapsize - robot_pos_y);
	return row_offset * 2 * mapsize + col_offset;
}


std::vector<float> convert_index(float mapsize, int index)
{
	int row = index / (2 * mapsize);
	int col = index % (int)(2 * mapsize);
	std::vector<float> pos;
	pos.push_back(col - mapsize);
	pos.push_back(mapsize - row);

	return pos;
}


void print_distance_left(node best_node, PARAM* param)
{
	float average_x = 0;
	float average_y = 0;
	for (int i = 0; i < param->N; i++)
	{
		average_x += best_node.robot_pos[i][0];
		average_y += best_node.robot_pos[i][1];
	}
	average_x /= param->N;
	average_y /= param->N;
	float distance_left = sqrt(pow(param->target_center[0] - average_x, 2) + pow(param->target_center[1] - average_y, 2));
	printf("Distance left is %f\n", distance_left);
	return;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////


Queue* initializeQps(PARAM* param, node h_start)
{
	Queue* qps = new Queue[QUEUE_SIZE];

	for (int i = 0; i < param->q_count; i++)
	{
		qps[i].iteration = 1;
	}



	for (int i = 0; i < param->q_count; i++)
	{
    qps[i].h_open.push_back(h_start);
	}

	return qps;
}


float h_calculate_H(node d_expanded, PARAM* d_param, int mode)
{
	float priority = 0;
	float priority_i;
	int i;
	float target_radius = d_param->target_radius;
	float target_center[2];
	target_center[0] = d_param->target_center[0];
	target_center[1] = d_param->target_center[1];
	float robot_radius = d_param->robot_radius;
	int N = d_param->N;
	for (int i = 0; i < N; i++)
	{
		priority_i = sqrt(pow(d_expanded.robot_pos[i][0] - target_center[0], 2)
			+ powf(d_expanded.robot_pos[i][1] - target_center[1], 2));
		priority_i -= target_radius - robot_radius;
		priority_i = max(priority_i, (float) 0.0);
		priority += priority_i;
	}
	if (mode == 0) return priority;
	else if (mode == 1) {
		float dist_to_ob = 0;
		float dist_to_ob_i = 0;
		for (int i = 0; i < d_param->N; i++)
		{
			dist_to_ob_i = 0;
			dist_to_ob_i = sqrt(powf(d_expanded.robot_pos[i][0] - d_param->obstacle_pos[0][0], 2) + powf(d_expanded.robot_pos[i][1] - d_param->obstacle_pos[0][1], 2)) - d_param->obstacle_pos[0][2];
			for (int j = 0; j < d_param->M; j++)
			{
				dist_to_ob_i = fminf(dist_to_ob_i, sqrt(powf(d_expanded.robot_pos[i][0] - d_param->obstacle_pos[j][0], 2) + powf(d_expanded.robot_pos[i][1] - d_param->obstacle_pos[j][1], 2)) - d_param->obstacle_pos[0][2]);
			}
			dist_to_ob_i = dist_to_ob_i - 3;
			dist_to_ob_i = fmaxf(dist_to_ob_i, 0);
			dist_to_ob += dist_to_ob_i;
		}


		//printf("priority, additional part (%f, %f)\n", priority, powf(3, -(dist_to_ob - 5)));


		return priority + dist_to_ob;
	}

}

void updateQueues(Queue* qps, node curr, PARAM* param, int queue_i)
{
	curr.F = curr.G + param->H * h_calculate_H(curr, param, 0);
	float queue_0_F = curr.F;
  //printf("UpdateQueues>>>> curr.F = %f, curr.behaviorIdx = %llu\n", curr.F, curr.behaviorIdx);

	qps[0].h_open.push_back(curr);
  int index = qps[0].h_open.size() - 1;
  //printf("Just to make sure UpdateQueus>>> curr.F = %f, curr.behaviorIdx = %llu\n", 
  //       qps[0].h_open[index].F, qps[0].h_open[index].behaviorIdx);
	for (int i = 1; i < QUEUE_SIZE; i++)
	{
		curr.F = curr.G + param->H * h_calculate_H(curr, param, i);
		//printf("curr_i_F = %f, cuff_0_F = %f\n", curr.F, queue_0_F);
		if (curr.F <= param->H2 * queue_0_F) qps[i].h_open.push_back(curr);
	}
	return;
}


void expandStates(Queue* qps, PARAM* param, node* best_node_p, node* best_attempt, int queue_i)
{
	PARAM* d_param; /* device parameters */
	hipMalloc(&d_param, sizeof(PARAM));
	hipMemcpy(d_param, param, sizeof(PARAM), hipMemcpyHostToDevice);

	node *d_open; /* device open nodes */
	node *d_expanded; /* device expanded nodes */

					  /* Dequeue from h_open */
	int iteration = qps[queue_i].iteration;
	printf("Inside ExpandStates....iteration is %d\n", iteration);

	int h_open_size = qps[queue_i].h_open.size();
	int real_copies = (h_open_size > ARRAY_SIZE) ? ARRAY_SIZE : h_open_size;
	node* h_open_array = new node[real_copies]; /* temporary data used for hipMemcpy of h_open */
	node* h_expanded = new node[real_copies * DIR]; /* data used for retrieving expanded nodes data; Also used for backtracking */

	printf("For %d, real_copies %d\n", queue_i, real_copies);

	std::copy(qps[queue_i].h_open.begin(), qps[queue_i].h_open.begin() + real_copies, h_open_array);
	/* Erase the same nodes from all queues */
	qps[queue_i].h_open.erase(qps[queue_i].h_open.begin(), qps[queue_i].h_open.begin() + real_copies);
	/* TODO: Make sure this portion of code is correct!!!! */


	//////////////
	for (int queue_j = 0; queue_j < param->q_count; queue_j++)
	{
		if (queue_i == queue_j) continue;
		std::vector<int> erase_indices;
		for (int i = 0; i < real_copies; i++) {
			for (int j = 0; j < qps[queue_j].h_open.size(); j++)
			{
				if (h_open_array[i].behaviorIdx == qps[queue_j].h_open[j].behaviorIdx) {
					erase_indices.push_back(j);
					break;
				}
			}
		}

		std::sort(erase_indices.begin(), erase_indices.end());

		for (int k = erase_indices.size() - 1; k >= 0; k--)
		{
			//printf("erase_indices[%d] is %d\n", k, erase_indices[k]);

			qps[queue_j].h_open.erase(qps[queue_j].h_open.begin() + erase_indices[k]);
		}
		
	}


	/* Copy necessary data to device memory */
	hipMalloc(&d_open, sizeof(node) * real_copies);
	hipMemcpy(d_open, h_open_array, sizeof(node) * real_copies, hipMemcpyHostToDevice);
	hipMalloc(&d_expanded, sizeof(node) * real_copies * DIR);


	/* Allocate proper blocks and threads */
	const dim3 blockSize(DIR, param->N, 1);
	const dim3 gridSize(real_copies, 1, 1);
	//const dim3 blockSize(real_copies, 1, 1);
	//const dim3 gridSize(dir, param->N, 1);

	/* Run the GPU code */
	k_expandStates << < gridSize, blockSize >> >(d_expanded, d_open, d_param, DIR, iteration, real_copies, queue_i);

	/* Copy back from GPU to CPU */
	hipMemcpy(h_expanded, d_expanded, sizeof(node) * real_copies * DIR, hipMemcpyDeviceToHost);

	/* Update open list with expanded nodes and update best_node */
	for (int ind = 0; ind < real_copies * DIR; ind++)
	{ 
    /* Debuggin */
   if (h_expanded[ind].isEmpty == 0 && h_expanded[ind].reached_destination == 0)
	 {
      //printf("After Kernel>>>>> [%d].F = %f, .behaviorIdx = %llu\n", 
      //      ind, h_expanded[ind].F, h_expanded[ind].behaviorIdx);
		
	
			node curr = h_expanded[ind];
			float distance_to_goal = curr.F - curr.G;
  		if(distance_to_goal < best_attempt->F - best_attempt->G) memcpy(best_attempt, &curr, sizeof(node));
			updateQueues(qps, curr, param, queue_i);
		}
		else if (h_expanded[ind].isEmpty == 0 && h_expanded[ind].reached_destination == 1
			&& h_expanded[ind].G < best_node_p->G) {
			memcpy(best_node_p, &h_expanded[ind], sizeof(node));
			printf("UPDATING BEST NODE!! cost is %f, sequence_numel is %d\n", best_node_p->G, best_node_p->sequence_numel);
			printf("printing sequence...\n");
			for (int i = 0; i < best_node_p->sequence_numel; i++)
			{
				int index = best_node_p->behaviorIndices[i];
				//printf("hello world????\n");
				printf("%s, ", behavior_array[index]);
				//printf("%d ", index);
			}
			printf("\n");
		}
	}

	for (int i = 0; i < param->q_count; i++)
	{
		printf("Queue%d size is %d\n", i, qps[i].h_open.size());
	}


	/* Sort the open_list */
	for (int i = 0; i < QUEUE_SIZE; i++)
	{
		std::sort(qps[i].h_open.begin(), qps[i].h_open.end(), [](node left, node right) {return left.F < right.F; });
	}



	/* Prepare for next iteration */
	qps[queue_i].iteration++;
	delete(h_open_array);
	hipFree(d_expanded);
	hipFree(d_open);
	hipFree(d_param);

}

__global__
void k_noSMHA(POS* d_poses, node* d_result, PARAM* d_param, int* d_sequence_end_indices)
{
	int robot_index = threadIdx.x;
	/* Error checking */
	if (robot_index >= d_param->N) return;
	
	if (robot_index == 0) {
		memcpy(d_poses[0].robot_pos, d_param->robot_pos, sizeof(float)*d_param->N * 3);
		printf("Inside k_noSMHA!!!!\n");
		printf("SEQUENCE NUMEL IS %d\n", d_result->sequence_numel);
	}
	__syncthreads();
	
	/* Expand the nodes */
	float dt = d_param->dt;
	float ti = d_param->ti;
	float dT = d_param->dT;
	float tf = d_param->tf;
	int steps = (int)dT / dt;
	float dstart = ti;
	float dend = 0;

	int sequence_count = d_result->sequence_numel;

	node d_local;
	memcpy(&d_local, d_result, sizeof(node));
	int d_poses_index = 0;
	for (int i = 0; i < sequence_count; i++)
	{
		dstart = ti + dstart + dend;
		dend = d_param->time_array[i];
		if(i == 0) steps = (int) d_param->time_array[i] / dt;
		else if(i == sequence_count - 1) steps = (int) (tf - d_param->time_array[i - 1]) / dt;
		else steps = (int) (d_param->time_array[i] - d_param->time_array[i-1]) / dt;
		for (int j = 1; j <= steps; j++)
		{
			/* Forward kinematics -> Save it to POS* -> Update d_local */
			func[d_result->behaviorIndices[i]](d_result, &d_local, robot_index, dt);
			__syncthreads();
		
			if(d_valid_poses(*d_result, d_param) == 0) { d_result->isEmpty = 1;}
			d_result->G = d_calculate_G(*d_result, d_local, d_param, 0);
			if (robot_index == 0) {
				memcpy(d_poses[d_poses_index].robot_pos, d_result->robot_pos, 
				sizeof(float) * d_param->N * 3);
			}
			__syncthreads();
			memcpy(&d_local, d_result, sizeof(node));
			d_poses_index++;
		}
		d_sequence_end_indices[i] = d_poses_index - 1;
	}

  d_result->reached_destination = d_target_reached(*d_result, d_param);



	return;
}


void noSMHAstar(PARAM* param, RETURN* return_1, node result_node)
{
	/* Only for debugging purposes need to make sure that all the info is delivered correctly */
	printf("Inside noSMHAstar.....\n");
	printf("number of switch times given.... %d\n", param->time_array_count);
	for(int i = 0; i < param->time_array_count; i++)
	{
		printf("%f ", param->time_array[i]);
	}
	printf("\n");

	printf("checking result node also.... sequence_numel is %d\n", result_node.sequence_numel);
	for(int i = 0; i < param->time_array_count + 1; i++)
	{
		printf("%s ", behavior_array[result_node.behaviorIndices[i]]);
	}
	printf("\n");



	PARAM* d_param; /* device parameters */
	gpuErrchk(hipMalloc(&d_param, sizeof(PARAM)));
	gpuErrchk(hipMemcpy(d_param, param, sizeof(PARAM), hipMemcpyHostToDevice));

	node* d_result;
	gpuErrchk(hipMalloc(&d_result, sizeof(node)));
	gpuErrchk(hipMemcpy(d_result, &result_node, sizeof(node), hipMemcpyHostToDevice));


	int seq_n = result_node.sequence_numel;
	int h_result_size = param->tf / param->dt + 1;
	printf("h_result_size is ... %d\n", h_result_size);
	POS* h_poses = new POS[h_result_size];
	POS* d_poses;
	gpuErrchk(hipMalloc(&d_poses, sizeof(POS) * h_result_size));

	
	/* Allocate space on device for sequence_end_indices array, cost_of_path, is_valid_path */
	float cost_of_path = 0;
	int is_valid_path = 0;
	int* h_sequence_end_indices = new int[param->time_array_count + 1];
	int* d_sequence_end_indices;
	gpuErrchk(hipMalloc(&d_sequence_end_indices, sizeof(int) * (param->time_array_count + 1)));



	const dim3 gridSize(1, 1, 1);
	const dim3 blockSize(param->N, 1, 1);
	k_noSMHA << <gridSize, blockSize >> >(d_poses, d_result, d_param, d_sequence_end_indices);

	/* Copy back from GPU to CPU */
	hipMemcpy(h_poses, d_poses, sizeof(POS) * h_result_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_sequence_end_indices, d_sequence_end_indices, 
		   sizeof(int) * (param->time_array_count + 1), hipMemcpyDeviceToHost);

	node* h_result = new node[1];
	hipMemcpy(h_result, d_result, sizeof(node), hipMemcpyDeviceToHost);

	
	printf("returned from kernel...\n");
	printf("printing all necessary information to check that kernel returned correctly\n");
	printf("Cost of path(G) = %f\n",h_result->G);
	if(h_result->isEmpty == 0) printf("PATH IS VALID\n");
 	else printf("PATH IS INVALID\n");

	/* Copy all the necessary information back to return struct */
  return_1->cost_of_path = h_result->G;
	if(h_result->isEmpty) return_1->is_valid_path = 0;
	else return_1->is_valid_path = 1;
	return_1->is_complete = h_result->reached_destination;

	for(int i = 0; i < h_result_size; i++)
	{
		return_1->robot_positions.push_back(h_poses[i]);
	}
	for(int i = 0; i < seq_n; i++)
	{
		return_1->sequence_end_indices.push_back(h_sequence_end_indices[i]);
		return_1->sequence_string_array.push_back(behavior_array[result_node.behaviorIndices[i]]);
	}

	printf("making sure that return struct has necessary info...\n");
	printf("return_1->cost_of_path = %f\n", return_1->cost_of_path);
	printf("return-1->is_valid_path = %d\n", (int) return_1->is_valid_path);
  for(int i = 0; i < seq_n; i++)
  {
    printf("return_1->sequence_end_indices[%d] = %d\n", 
            i, return_1->sequence_end_indices[i]);
  }


	return;
}


/* returns @ string of behavior sequence (for now) behavior__behavior__behavior__....behavior*/
node SMHAstar(PARAM* param, node h_start)
{
	clock_t start = clock();
	cout << "Starting IMHA star " << endl;

	Queue* qps = initializeQps(param, h_start);
	node result;
	result.isEmpty = 1;
	result.F = numeric_limits<float>::max();

	node* best_node_p = new node[1];
	best_node_p->reached_destination = 0;
	best_node_p->isEmpty = 1;
	best_node_p->F = numeric_limits<float>::max();
	best_node_p->G = numeric_limits<float>::max();
	best_node_p->N = param->N;

	node best_attempt; //criteria is distance to destination
	best_attempt.isEmpty = 1;
	best_attempt.F = numeric_limits<float>::max();
	best_attempt.G = 0;
	best_attempt.N = param->N;

	//convert double to float for time_arra
	while (!qps[0].h_open.empty())
	{
		/* Exit if exceeded the amount of time */
		clock_t end = clock();
		float time_elapsed = float(end - start);

		if (time_elapsed > 20000000) //10 sec
		{
			printf("Exceeded time limit of %f (ms)", time_elapsed);
			if(result.isEmpty && best_node_p->isEmpty) 
			memcpy(&result, &best_attempt, sizeof(node));
			else if(result.isEmpty)
			memcpy(&result, best_node_p, sizeof(node));

			return result;
		}
  		
		for (int queue_i = 1; queue_i < param->q_count; queue_i++)
		{
			//Since we will only have 2 queues, I will do some hardcoding
			//If there are no elements in queue1, then just expand queue0
			if (qps[queue_i].h_open.empty()) {
        /* For debugging purposes */
       /* printf("Printing the whole queue0\n");
        for(int i = 0; i < qps[0].h_open.size(); i++)
        {
          //printf("[%d].F = %f, .sequence_numel = %d .behaviorIdx = %llu \n", i, 
          //  qps[0].h_open[i].F, qps[0].h_open[i].sequence_numel, 
				//		qps[0].h_open[i].behaviorIdx);
        }                                 //}				
				*/

        node minKey_0 = qps[0].h_open[0];
				
				printf("minKey_0.F %f \n", minKey_0.F);
				printf("best_i.G %f\n", best_node_p->G);
				

				if (best_node_p->G <= minKey_0.F)
				{
					if (best_node_p->G < numeric_limits<float>::max())
					{
						printf("DONE WITH SEARCH!! NOW PRINTING RESULTS\n");
						printf("Current best node cost %f, sequence_numel is %d\n", best_node_p->G, best_node_p->sequence_numel);
						print_distance_left(*best_node_p, param);

						printf("printing sequence...\n");
						for (int i = 0; i < best_node_p->sequence_numel; i++)
						{
							int index = best_node_p->behaviorIndices[i];
							printf("%s, ", behavior_array[index]);
	  	 			}
						printf("\n");

						memcpy(&result, best_node_p, sizeof(node));
						clock_t end = clock();
						float time_elapsed = float(end - start);
						cout << "Time to calculate the route (ms): " << time_elapsed << endl;
						printf("inside smha!!! result sequence_numel!!!%d\n", result.sequence_numel);
						return result;
					}
				}
				else
				{
					expandStates(qps, param, best_node_p, &best_attempt, 0);
				}
			}
			else {
				node minKey_i = qps[queue_i].h_open[0];
				node minKey_0 = qps[0].h_open[0];

				/* Debugging purposes.... */

				printf("minKey_i.F, minKey_0.F %f %f \n", minKey_i.F, minKey_0.F);
				printf("best_i.G %f\n", best_node_p->G);

				if (minKey_i.F <= param->H2 * minKey_0.F)
				{
					printf("QUEUE %d!!!\n", queue_i);
					if (best_node_p->G <= minKey_i.F)
					{
						if (best_node_p->G < numeric_limits<float>::max())
						{
							//return path pointed by bpi(best_node)
							printf("DONE WITH SEARCH!! NOW PRINTING RESULTS\n");
							printf("Current best node cost %f, sequence_numel is %d\n", best_node_p->G, best_node_p->sequence_numel);

							printf("printing sequence...\n");
							for (int i = 0; i < best_node_p->sequence_numel; i++)
							{
								int index = best_node_p->behaviorIndices[i];
								printf("%s, ", behavior_array[index]);
							}
							printf("\n");

							print_distance_left(*best_node_p, param);

							memcpy(&result, best_node_p, sizeof(node));
							clock_t end = clock();
							float time_elapsed = float(end - start);
							cout << "Time to calculate the route (ms): " << time_elapsed << endl;
							printf("inside smha!!! result sequence_numel!!!%d\n", result.sequence_numel);
							return result;

						}
					}
					else {
						expandStates(qps, param, best_node_p, &best_attempt, queue_i);
					}
				}
				else
				{
					printf("queue 0!!!\n");
					if (best_node_p->G <= minKey_0.F)
					{
						if (best_node_p->G < numeric_limits<float>::max())
						{
							printf("DONE WITH SEARCH!! NOW PRINTING RESULTS\n");
							printf("Current best node cost %f, sequence_numel is %d\n", best_node_p->G, best_node_p->sequence_numel);
							print_distance_left(*best_node_p, param);

							printf("printing sequence...\n");
							for (int i = 0; i < best_node_p->sequence_numel; i++)
							{
								int index = best_node_p->behaviorIndices[i];
								printf("%s, ", behavior_array[index]);
							}
							printf("\n");

							memcpy(&result, best_node_p, sizeof(node));
							clock_t end = clock();
							float time_elapsed = float(end - start);
							cout << "Time to calculate the route (ms): " << time_elapsed << endl;
							printf("inside smha!!! result sequence_numel!!!%d\n", result.sequence_numel);
							return result;
						}
					}
					else
					{
						expandStates(qps, param, best_node_p, &best_attempt, 0);
					}
				}
			}
			
		}
		printf("\n\n");
	}

	for (int queue_j = 0; queue_j < param->q_count; queue_j++)
	{
		if (best_node_p->isEmpty == 0 && best_node_p->G < result.G) memcpy(&result, best_node_p, sizeof(node));
	}


	if (result.isEmpty == 1) {
		printf("no route found, returning the best attempt\n");
		memcpy(&result, &best_attempt, sizeof(node));
	}
	else {
		printf("printing sequence...\n");
		for (int i = 0; i < result.sequence_numel; i++)
		{
			int index = result.behaviorIndices[i];
			printf("%s, ", behavior_array[index]);
		}
		printf("\n");
	}


	clock_t end = clock();
	float time_elapsed = float(end - start);
	cout << "Time to calculate the route (ms): " << time_elapsed << endl;
	return result;


}


__global__
void k_SAVE(POS* d_poses, node* d_result, node* d_start, PARAM* d_param)
{
	int robot_index = threadIdx.x;
	/* Error checking */
	if (robot_index >= d_param->N) return;

	if (robot_index == 0) memcpy(d_poses[0].robot_pos, d_param->robot_pos, sizeof(float)*d_param->N * 3);

	__syncthreads();

	/* Expand the nodes */
	float dt = d_param->dt;
	float ti = d_param->ti;
	float dT = d_param->dT;
	float tf = d_param->tf;
	int steps = (int)dT / dt;

	int sequence_count = d_result->sequence_numel;

	node d_local;
	memcpy(&d_local, d_start, sizeof(node));
	for (int i = 0; i < sequence_count; i++)
	{

		for (int j = 1; j <= steps; j++)
		{
			/* Forward kinematics -> Save it to POS* -> Update d_local */
			func[d_result->behaviorIndices[i]](d_start, &d_local, robot_index, dt);
			__syncthreads();
			if (robot_index == 0) memcpy(d_poses[i*steps + j].robot_pos, d_start->robot_pos, sizeof(float) * d_param->N * 3);
			__syncthreads();
			memcpy(&d_local, d_start, sizeof(node));
		}
	}


	return;

}

void SAVE_launch(node result_node, RETURN* return_1, PARAM* param)
{
	/* Open file and retrieve file id */

	printf("Printing out the sequence...\n");
	for(int i = 0; i < result_node.sequence_numel; i++)
	{
		printf("%s ", behavior_array[result_node.behaviorIndices[i]]);
	}
	printf("\n");

	PARAM* d_param; /* device parameters */
	gpuErrchk(hipMalloc(&d_param, sizeof(PARAM)));
	gpuErrchk(hipMemcpy(d_param, param, sizeof(PARAM), hipMemcpyHostToDevice));

	node* d_result;
	gpuErrchk(hipMalloc(&d_result, sizeof(node)));
	gpuErrchk(hipMemcpy(d_result, &result_node, sizeof(node), hipMemcpyHostToDevice));

	node h_start;
	h_start.isEmpty = 0;
	h_start.N = param->N;
	h_start.sequence_numel = 0;
	memcpy(&h_start.robot_pos, &param->robot_pos, sizeof(float)*ROBOT_MAX * 3);


	/* DEBUGGING!!!! */
	printf("making sure h_start has correct positions.....\n");
	for (int i = 0; i < param->N; i++)
	{
		printf("(%f %f %f) ", h_start.robot_pos[i][0], h_start.robot_pos[i][1], h_start.robot_pos[i][2]);
	}
	printf("\n");
	/* End of debugging.... */

	node* d_start;
	gpuErrchk(hipMalloc(&d_start, sizeof(node)));
	gpuErrchk(hipMemcpy(d_start, &h_start, sizeof(node), hipMemcpyHostToDevice));


	int steps = (int)param->dT / param->dt;
	int seq_n = result_node.sequence_numel;
	int h_result_size = steps* seq_n + 1;
	printf("steps, seq_n, %d, %d\n", steps, seq_n);
	printf("h_result_size is ... %d\n", h_result_size);
	POS* h_poses = new POS[h_result_size];
	POS* d_poses;
	gpuErrchk(hipMalloc(&d_poses, sizeof(POS) * h_result_size));




	const dim3 gridSize(1, 1, 1);
	const dim3 blockSize(param->N, 1, 1);
	k_SAVE << <gridSize, blockSize >> >(d_poses, d_result, d_start, d_param);

	/* Copy back from GPU to CPU */
	hipMemcpy(h_poses, d_poses, sizeof(POS) * h_result_size, hipMemcpyDeviceToHost);

	/* Now save everything onto the txt */
	cout << "saving everything in txt file" << endl;


	//output_f = fopen(output_fname.str().c_str(), "w");
	int times = 0;
	printf("H_RESULT_SIZE IS %d\n", h_result_size);
	for (int i = 0; i < h_result_size; i++)
	{
		for (int j = 0; j < param->N; j++)
		{
			//fprintf(output_f, (std::to_string(h_poses[i].robot_pos[j][0]) + " " + std::to_string(h_poses[i].robot_pos[j][1])
			//	+ " " + std::to_string(h_poses[i].robot_pos[j][2]) + "\n").c_str());
			times++;
		}
		//fprintf(output_f, "\n");
	}
	return;
}

void SMHAstar_wrapper(PARAM* param, RETURN* result_1)
{
  printf("Inside smhastar_wrapper\n");
	node h_start; //starting node

	h_start.isEmpty = 0;
	h_start.N = param->N;
	h_start.sequence_numel = 0;

	printf("Inside smhastar_wrapper, param->N = %d\n", param->N);


	std::copy(&param->robot_pos[0][0], &param->robot_pos[0][0] + param->N * 3, &h_start.robot_pos[0][0]);

	printf("Done copying robot_positions param->h_start\n");
	h_start.F = param->H * h_calculate_H1(h_start.robot_pos, param, h_start.N);
	h_start.G = 0;
	h_start.reached_destination = 0;
	
	node result_node = SMHAstar(param, h_start); //note this result_node might be simply the closest attempt to the goal

	//Need to reset the result_node robot position to initial 
	std::copy(&param->robot_pos[0][0], &param->robot_pos[0][0] + param->N * 3, 
		&result_node.robot_pos[0][0]);

	printf("checking the sequence before entering noSMHAstar...\n");
	printf("result_node sequence count is %d\n", result_node.sequence_numel);
	for(int i = 0; i < result_node.sequence_numel; i++)
	{
		printf("%s ", behavior_array[result_node.behaviorIndices[i]]);

	}
	printf("\n");

	noSMHAstar(param, result_1, result_node);
	return;

}








/////////////////////////////////////////////////////////////////////////////////////////////////////////

void initialize_parameters(PARAM* param, std::vector<float> time_array, std::vector<int>sequence_array, std::vector<uint8_t> fix_array)
{ 
  /*
	param->N = 16;
	param->M = 3; //2
	param->H = 1;
	param->mapsize = 20;
	param->ti = 0;
	param->dt = 0.1;
	param->tf = 50;
	param->dT = 5;
	param->target_center[0] = 10;
	param->target_center[1] = 10;
	param->target_radius = 7;
	param->robot_radius = 0.5;
	param->q_count = 2;
	param->H2 = 1;

	/ Initialize robot_pos /
	float bottom_left_x = -param->mapsize + 1;
	float bottom_left_y = -param->mapsize + 1;
	float width = (param->target_radius) * 2;
	float height = (param->target_radius) * 2;

	int i;
	for (i = 0; i < param->N; i++)
	{
		param->robot_pos[i][0] = bottom_left_x + width * ((float)(rand() % 1000)) / 1000;
		param->robot_pos[i][1] = bottom_left_y + height * ((float)(rand() % 1000)) / 1000;
		param->robot_pos[i][2] = remainder(((float)(rand() % 1000)) / 1000, 2.0*M_PI);
	}

	/ Initialize obstacle_pos /
	float obstacle_min_radius = 2;
	float obstacle_max_radius = 5;

	for (i = 0; i < param->M; i++)
	{
		param->obstacle_pos[i][0] = fmod(((float)(rand() % 1000)) / 10, (2 * param->mapsize)) - param->mapsize;
		param->obstacle_pos[i][1] = fmod(((float)(rand() % 1000)) / 10, (2 * param->mapsize)) - param->mapsize;
		param->obstacle_pos[i][2] = fmod(((float)(rand() % 1000)) / 10, obstacle_max_radius);
		if (param->obstacle_pos[i][2] < obstacle_min_radius) param->obstacle_pos[i][2] += obstacle_min_radius;
	}
  */
  
	param->time_array_count = time_array.size();
	int fix_count = 0;
	printf("Inside initialize_params....\n");
	printf("Length of time_array is %d, Length of sequence is %d\n", time_array.size(), sequence_array.size());

	for(int i = 0; i < time_array.size(); i++)
	{
		param->time_array[i] = time_array[i];
	}
	for(int i = 0; i < sequence_array.size(); i++)
	{
		param->sequence_array[i] = sequence_array[i];
		if((int) fix_array[i] == 1) fix_count++;
	}
  /*
  for(int i = 0; i < param->N; i++)
	{
		printf("robot %d pos = %f %f %f\n", i, param->robot_pos[i][0], param->robot_pos[i][1],
						param->robot_pos[i][2]);
	}

	for(int i = 0; i < param->M; i++)
	{
		printf("obstacle %d pos = %f %f %f\n", i, param->obstacle_pos[i][0], param->obstacle_pos[i][1],
					param->obstacle_pos[i][2]);
	}
  */
	param->fix_count = fix_count;
  printf("Returning from init params\n");
	return;
}


void fix_robot_positions(PARAM* param)
{
	param->N = 10;
	param->robot_pos[0][0] = -18;
	param->robot_pos[0][1] = -10;
	param->robot_pos[0][2] = 0;

	param->robot_pos[1][0] = -17;
	param->robot_pos[1][1] = -10;
	param->robot_pos[1][2] = 0;

	param->robot_pos[2][0] = -16;
	param->robot_pos[2][1] = -10;
	param->robot_pos[2][2] = 0;

	param->robot_pos[3][0] = -15;
	param->robot_pos[3][1] = -10;
	param->robot_pos[3][2] = 0;

	param->robot_pos[4][0] = -14;
	param->robot_pos[4][1] = -10;
	param->robot_pos[4][2] = 0;

	param->robot_pos[5][0] = -18;
	param->robot_pos[5][1] = -11;
	param->robot_pos[5][2] = 0;

	param->robot_pos[6][0] = -17;
	param->robot_pos[6][1] = -11;
	param->robot_pos[6][2] = 0;

	param->robot_pos[7][0] = -16;
	param->robot_pos[7][1] = -11;
	param->robot_pos[7][2] = 0;

	param->robot_pos[8][0] = -15;
	param->robot_pos[8][1] = -11;
	param->robot_pos[8][2] = 0;

	param->robot_pos[9][0] = -14;
	param->robot_pos[9][1] = -11;
	param->robot_pos[9][2] = 0;

	param->robot_pos[10][0] = -18;
	param->robot_pos[10][1] = -10;
	param->robot_pos[10][2] = 0;

	param->robot_pos[11][0] = -17;
	param->robot_pos[11][1] = -10;
	param->robot_pos[11][2] = 0;

	param->robot_pos[12][0] = -16;
	param->robot_pos[12][1] = -10;
	param->robot_pos[12][2] = 0;

	param->robot_pos[13][0] = -15;
	param->robot_pos[13][1] = -10;
	param->robot_pos[13][2] = 0;

	param->robot_pos[14][0] = -14;
	param->robot_pos[14][1] = -10;
	param->robot_pos[14][2] = 0;

	param->robot_pos[15][0] = -18;
	param->robot_pos[15][1] = -9;
	param->robot_pos[15][2] = 0;

}

void fix_obstacle_positions(PARAM* param)
{
	param->M = 1;
	param->obstacle_pos[0][0] = 0;
	param->obstacle_pos[0][1] = 0;
	param->obstacle_pos[0][2] = 5;

	param->obstacle_pos[1][0] = -10;
	param->obstacle_pos[1][1] = 10;
	param->obstacle_pos[1][2] = 7;
}





void initialize_result(node* result_node, PARAM* param)
{
	result_node->isEmpty = 1;
	result_node->N = param->N;
	result_node->sequence_numel = param->time_array_count + 1;
	memcpy(&result_node->robot_pos[0][0], &param->robot_pos[0][0], sizeof(float) * param->N * 3);
	result_node->F = param->H * h_calculate_H1(result_node->robot_pos, param, result_node->N);
	result_node->G = 0;
	result_node->reached_destination = 0;
	memcpy(&result_node->behaviorIndices, &param->sequence_array, sizeof(int)*SEQ_MAX);
	return;
}

RETURN testmain(PARAM* param, int isAided, std::vector<float> time_array, std::vector<int> sequence_array, std::vector<uint8_t> isFixed)
{
	printf("starting\n");
	RETURN return_1;

	initialize_parameters(param, time_array, sequence_array, isFixed);

	if(isAided) SMHAstar_wrapper(param, &return_1);
	else {
		node result_node;
		initialize_result(&result_node, param);
	
		noSMHAstar(param, &return_1, result_node);
		//printf("after returning from noSMHAstar function....\n");
		//printf("return_1 cost_of_path = %f\n", return_1.cost_of_path);
	}

	return return_1;
}
