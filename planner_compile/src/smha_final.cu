#include "hip/hip_runtime.h"
/* Implement SMHA - this one works!!!! */
/* 9/8/2017 making modifications for linux version */


#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <queue>
#include <string>
#include <iterator>
#include <fstream>
#include <sstream>
#include <algorithm>

#define _USE_MATH_DEFINES
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <limits>
#include <vector>
#include <set>
#include <algorithm>
#include <map>

#include "math.h"
#include "hip/hip_math_constants.h"
#include "smha_headers.hpp"

//////////////////////////////////////////////////////////////////////////////////////////////////////////

using namespace std;

char *behavior_array[DIR] = { "rendezvous", "antirendezvous", "flock_east", "flock_north", "flock_west", "flock_south", "line_x", "line_y" };

std::vector<string> behavior_array_display = {"Rendezvous", "Antirendezvous", "Flock East", "Flock North", "Flock West", "Flock South", "Line X", "Line Y"};

/* Error Checking..... */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////


/* DEVICE FUNCTIONS (BEHAVIOR and KERNEL) */

__device__
float d_wrapToPi(float input)
{
	float result = fmodf(input + HIP_PI_F, 2 * HIP_PI_F);
	result += (result < 0) * 2 * HIP_PI_F;
	result -= HIP_PI_F;
	return result;
}

__forceinline__ __device__
void updateSeenMap(node* ctx, int i)
{

  int grid_x = (int) rintf(ctx->robot_pos[i][0] + MAPSIZE);
  int grid_y = (int) rintf(ctx->robot_pos[i][1] + MAPSIZE);
  grid_x = fmaxf(grid_x, 0);
  grid_x = fminf(grid_x, MAPSIZE * 2 - 1);
  grid_y = fmaxf(grid_y, 0);
  grid_y = fminf(grid_y, MAPSIZE * 2 - 1);
  if(ctx->seen_map.seen[grid_y][grid_x] == 0)
  {
  	printf("Seen update (%f, %f), (%d, %d)\n", 
  		ctx->robot_pos[i][1], ctx->robot_pos[i][0], 
  		(int) rintf(ctx->robot_pos[i][1]), (int) rintf(ctx->robot_pos[i][0]));
    ctx->seen_map.seen[grid_y][grid_x] = 1;
    atomicAdd(&ctx->seen_map.seen_count, 1);
  }
  return;
}


__device__
void d_robot_model_i(node* current, int i, float u_v, float u_w, float dt)
{
	float max_linear_velocity = 8; //Doubled max linear, angular velocity by 2
	float max_angular_velocity = HIP_PI_F / 8;
	u_v = fmaxf(-max_linear_velocity, fminf(max_linear_velocity, u_v));
	u_w = fmaxf(-max_angular_velocity, fminf(max_angular_velocity, u_w));
	current->robot_pos[i][0] = current->robot_pos[i][0] + u_v*cosf(current->robot_pos[i][2])*dt;
	current->robot_pos[i][1] = current->robot_pos[i][1] + u_v*sinf(current->robot_pos[i][2])*dt;
	current->robot_pos[i][2] = d_wrapToPi(current->robot_pos[i][2] + u_w*dt);
	updateSeenMap(current, i);
	return;
}


__device__
void d_formation_control(node* future, node* current, int i, float formation[][2], float dt)
{
	float connectivity_radius = 20.0f;
	float gain_v = 4.0f;
	float gain_w = 4.0f;
	float position_i[2], position_j[2];
	float heading_i, heading_j;
	float v[2], d[2], dv[2], b[2];
	float w, n, dtheta;
	float norm_d;
	float u_v, u_w;

	int N = current->N;
	position_i[0] = current->robot_pos[i][0];
	position_i[1] = current->robot_pos[i][1];
	heading_i = current->robot_pos[i][2];

	v[0] = 0;
	v[1] = 0;
	w = 0;
	n = 0;

	for (int j = 0; j < N; j++)
	{
		if(i == j) continue;
		position_j[0] = current->robot_pos[j][0];
		position_j[1] = current->robot_pos[j][1];
		heading_j = current->robot_pos[j][2];

		d[0] = position_j[0] - position_i[0];
		d[1] = position_j[1] - position_i[1];
		norm_d = sqrtf(powf(d[0], 2) + powf(d[1], 2));
		dtheta = 0;

		if (norm_d < connectivity_radius)
		{
			dv[0] = d[0] - (formation[j][0] - formation[i][0]);
			dv[1] = d[1] - (formation[j][1] - formation[i][1]);
			n += 1;
		}
		else{
			dv[0] = 0;
			dv[1] = 0;
		}
		v[0] += dv[0];
		v[1] += dv[1];

	}
	v[0] /= (n + 1);
	v[1] /= (n + 1);

	float tmp = atan2f(v[1], v[0]);
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	dtheta = tmp - heading_i;

	tmp = atan2f(sinf(dtheta), cosf(dtheta));
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	w = w + tmp;

	b[0] = cosf(heading_i);
	b[1] = sinf(heading_i);
	u_v = gain_v * (v[0] * b[0] + v[1] * b[1]);
	u_w = gain_w * w;

	d_robot_model_i(future, i, u_v, u_w, dt);

}



__device__
void d_line_x(node* future, node *current, int i, float dt)
{
	float formation[ROBOT_MAX][2];
	for(int i = 0; i < ROBOT_MAX; i++)
	{
		formation[i][0] = 2*(i + 1);
		formation[i][1] = 0;
	}

	d_formation_control(future, current, i, formation, dt);
}

__device__
void d_line_y(node* future, node *current, int i, float dt)
{
	float formation[ROBOT_MAX][2];
	for(int i = 0; i < ROBOT_MAX; i++)
	{
		formation[i][0] = 0;
		formation[i][1] = 2 * (i + 1);
	} 

	d_formation_control(future, current, i, formation, dt);
}



__device__
void d_antirendezvous(node* future, node *current, int i, float dt)
{
	//printf("Inside rendezvous\n");
	float connectivity_radius = 20;
	float gain_v = 4;
	float gain_w = 4;
	int N = current->N;
	float v[2], dv[2], b[2];
	float w, n;
	float position_i[2], position_j[2], d[2];
	float heading_i;
	float dtheta;
	float u_v, u_w;
	position_i[0] = current->robot_pos[i][0];
	position_i[1] = current->robot_pos[i][1];
	heading_i = current->robot_pos[i][2];
	v[0] = 0;
	v[1] = 0;
	w = 0;
	n = 0;
	for (int j = 0; j<N; j++)
	{
		if (i == j) continue;
		position_j[0] = current->robot_pos[j][0];
		position_j[1] = current->robot_pos[j][1];
		//heading_j = current->robot_pos[j][2];
		d[0] = position_j[0] - position_i[0];
		d[1] = position_j[1] - position_i[1];
		float norm_d = sqrtf(powf(d[0], 2) + powf(d[1], 2));
		if (norm_d < connectivity_radius)
		{
			dv[0] = -d[0];
			dv[1] = -d[1];
			n = n + 1;
		}
		else
		{
			dv[0] = 0;
			dv[1] = 0;
		}
		v[0] = v[0] + dv[0];
		v[1] = v[1] + dv[1];
	}
	v[0] = v[0] / (n + 1);
	v[1] = v[1] / (n + 1);
	float tmp = atan2f(v[1], v[0]);
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	dtheta = tmp - heading_i;
	w = atan2f(sinf(dtheta), cosf(dtheta));
	w += 2 * HIP_PI_F * (w == -HIP_PI_F);
	b[0] = cosf(heading_i);
	b[1] = sinf(heading_i);
	u_v = gain_v * (v[0] * b[0] + v[1] * b[1]);
	u_w = gain_w * w;
	d_robot_model_i(future, i, u_v, u_w, dt);
	return;
}


__device__
void d_rendezvous(node* future, node *current, int i, float dt)
{
	//printf("Inside rendezvous\n");
	float connectivity_radius = 20;
	float gain_v = 4;
	float gain_w = 4;
	int N = current->N;
	float v[2], dv[2], b[2];
	float w, n;
	float position_i[2], position_j[2], d[2];
	float heading_i;
	float dtheta;
	float u_v, u_w;
	position_i[0] = current->robot_pos[i][0];
	position_i[1] = current->robot_pos[i][1];
	heading_i = current->robot_pos[i][2];
	v[0] = 0;
	v[1] = 0;
	w = 0;
	n = 0;
	for (int j = 0; j<N; j++)
	{
		if (i == j) continue;
		position_j[0] = current->robot_pos[j][0];
		position_j[1] = current->robot_pos[j][1];
		//heading_j = current->robot_pos[j][2];
		d[0] = position_j[0] - position_i[0];
		d[1] = position_j[1] - position_i[1];
		float norm_d = sqrtf(powf(d[0], 2) + powf(d[1], 2));
		if (norm_d < connectivity_radius)
		{
			dv[0] = d[0];
			dv[1] = d[1];
			n = n + 1;
		}
		else
		{
			dv[0] = 0;
			dv[1] = 0;
		}
		v[0] = v[0] + dv[0];
		v[1] = v[1] + dv[1];
	}
	v[0] = v[0] / (n + 1);
	v[1] = v[1] / (n + 1);
	float tmp = atan2f(v[1], v[0]);
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	dtheta = tmp - heading_i;
	w = atan2f(sinf(dtheta), cosf(dtheta));
	w += 2 * HIP_PI_F * (w == -HIP_PI_F);
	b[0] = cosf(heading_i);
	b[1] = sinf(heading_i);
	u_v = gain_v * (v[0] * b[0] + v[1] * b[1]);
	u_w = gain_w * w;
	d_robot_model_i(future, i, u_v, u_w, dt);
	return;
}

__device__
void d_flock_biased(node* future, node *current, int i, float* bias, float dt)
{
	float min_linear_velocity = 1;
	float repulsion_radius = 5;
	float alignment_radius = 10;
	float attraction_radius = 20;
	float gain_v = 4;
	float gain_w = 4;
	float position_i[2], position_j[2];
	float heading_i, heading_j;
	float v[2], d[2], dv[2], b[2];
	float w, n, dtheta;
	float norm_d;
	float u_v, u_w;
	int N = current->N;
	position_i[0] = current->robot_pos[i][0];
	position_i[1] = current->robot_pos[i][1];
	heading_i = current->robot_pos[i][2];
	v[0] = 0;
	v[1] = 0;
	w = 0;
	n = 0;

	for (int j = 0; j < N; j++)
	{
		if (i == j) continue;
		position_j[0] = current->robot_pos[j][0];
		position_j[1] = current->robot_pos[j][1];
		heading_j = current->robot_pos[j][2];
		n = n + 1;
		d[0] = position_j[0] - position_i[0];
		d[1] = position_j[1] - position_i[1];
		norm_d = sqrtf(powf(d[0], 2) + powf(d[1], 2));
		dtheta = 0;
		if (norm_d < repulsion_radius)
		{
			dv[0] = -d[0] / powf(norm_d, 2);
			dv[1] = -d[1] / powf(norm_d, 2);
			float tmp = atan2f(dv[1], dv[0]);
			tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
			dtheta = tmp - heading_i;
		}
		else if (norm_d < alignment_radius)
		{
			dv[0] = 0;
			dv[1] = 0;
			dtheta = heading_j - heading_i;
		}
		else if (norm_d < attraction_radius)
		{
			dv[0] = d[0];
			dv[1] = d[1];
			float tmp = atan2f(dv[1], dv[0]);
			tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
			dtheta = tmp - heading_i;
		}
		else
		{
			dv[0] = 0;
			dv[1] = 0;
			dtheta = 0;
			n = n - 1;
		}
		v[0] += dv[0];
		v[1] += dv[1];
		float tmp = atan2f(sinf(dtheta), cosf(dtheta));
		tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
		w = w + tmp;
	}
	v[0] = v[0] / (n + 1) + bias[0]; //incorporated Sasanka's line 51 and 54 at once
	v[1] = v[1] / (n + 1) + bias[1];
	w = d_wrapToPi(w / (n + 1));

	float tmp = atan2f(bias[1], bias[0]);
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	dtheta = tmp - heading_i;

	tmp = atan2f(sinf(dtheta), cosf(dtheta));
	tmp += 2 * HIP_PI_F * (tmp == -HIP_PI_F);
	w = w + tmp;

	b[0] = cosf(heading_i);
	b[1] = sinf(heading_i);
	u_v = gain_v * (v[0] * b[0] + v[1] * b[1]);
	u_w = gain_w * w;
	u_v = fmaxf(min_linear_velocity, u_v);

	d_robot_model_i(future, i, u_v, u_w, dt);
	return;
}


__device__
void d_flocking(node* future, node* current, int i, float dt)
{
	float bias[2] = { 0, 0 };
	d_flock_biased(future, current, i, bias, dt);
	return;
}


__device__
void d_flock_east(node* future, node* current, int i, float dt)
{
	float direction[2] = { 1, 0 };
	float gain = 1.0;
	float bias[2];
	bias[0] = gain * direction[0];
	bias[1] = gain * direction[1];
	d_flock_biased(future, current, i, bias, dt);
	return;
}


__device__
void d_flock_north(node* future, node* current, int i, float dt)
{
	float direction[2] = { 0, 1 };
	float gain = 1.0;
	float bias[2];
	bias[0] = gain * direction[0];
	bias[1] = gain * direction[1];
	d_flock_biased(future, current, i, bias, dt);
	return;
}

__device__
void d_flock_west(node* future, node* current, int i, float dt)
{
	float direction[2] = { -1, 0 };
	float gain = 1.0;
	float bias[2];
	bias[0] = gain * direction[0];
	bias[1] = gain * direction[1];
	d_flock_biased(future, current, i, bias, dt);
	return;
}

__device__
void d_flock_south(node* future, node* current, int i, float dt)
{
	float direction[2] = { 0, -1 };
	float gain = 1.0;
	float bias[2];
	bias[0] = gain * direction[0];
	bias[1] = gain * direction[1];
	d_flock_biased(future, current, i, bias, dt);
	return;
}

__device__
void d_move_stop(node* future, node* current, int i, float dt)
{
	return;
}


typedef void(*op_func) (node*, node*, int, float);
__device__ op_func func[DIR] = { d_rendezvous, d_antirendezvous, d_flock_east, d_flock_north, d_flock_west, d_flock_south, d_line_x, d_line_y };
__device__ char *d_behavior_array[DIR] = { "rendezvous", "antirendezvous", "flock_east", "flock_north", "flock_west", "flock_south", "line_x", "line_y" };


/* returns@ 0 is target is not reached; else 1 */
__device__
int d_target_reached(node curr_node, PARAM* param)
{
	int N = curr_node.N;
	float robot_radius = param->robot_radius;
	float target_radius = param->target_radius;
	/* L2norm(robot_pos(i) - target_center) */
	float robot_x, robot_y;
	float target_x = param->target_center[0];
	float target_y = param->target_center[1];
	float distance;
	for (int i = 0; i < N; i++)
	{
		robot_x = curr_node.robot_pos[i][0];
		robot_y = curr_node.robot_pos[i][1];
		distance = sqrt(powf(robot_x - target_x, 2) + powf(robot_y - target_y, 2));
		if (distance >= target_radius - robot_radius) return 0;
	}
	return 1;
}


/* returns@ 0 is not valid; else 1 */
__device__
int d_valid_poses(node curr_node, PARAM* param)
{
	int i, j;
	int N = curr_node.N;
	int M = param->M;
	float robot_x, robot_y;
	float robot_radius = param->robot_radius;
	float mapsize = param->mapsize;
	float safety_bounds = 0;
	/* check if position is off the map */
	for (i = 0; i< N; i++)
	{
		robot_x = curr_node.robot_pos[i][0];
		robot_y = curr_node.robot_pos[i][1];
		if (robot_x > mapsize || robot_y > mapsize ||
			robot_x < -mapsize || robot_y < -mapsize) {
			return 0;
		}
	}
	/* check if collided with obstacle */
	float obs_x, obs_y, obs_r;
	float distance;
	for (i = 0; i < N; i++)
	{
		robot_x = curr_node.robot_pos[i][0];
		robot_y = curr_node.robot_pos[i][1];
		for (j = 0; j < M; j++)
		{
			obs_x = param->obstacle_pos[j][0];
			obs_y = param->obstacle_pos[j][1];
			obs_r = param->obstacle_pos[j][2];
			distance = sqrt(powf(robot_x - obs_x, 2) + powf(robot_y - obs_y, 2));
			if (distance < obs_r + robot_radius + safety_bounds) {
				return 0;
			}
		}
	}
	return 1;
}




__device__
float d_calculate_G(node d_expanded, node d_open, PARAM* d_param, int mode)
{


	float final_G = d_open.G;
  float N = d_open.N;
	for (int i = 0; i < N; i++)
	{
		final_G += sqrtf(powf(d_expanded.robot_pos[i][0] - d_open.robot_pos[i][0], 2) +
			powf(d_expanded.robot_pos[i][1] - d_open.robot_pos[i][1], 2));
	}
	return final_G;
}



/* calculate the heuristic between curr node and the destination */
/* cost_to_go is distance left: cost_to_go = sum(max(pdist2(poses_next(:,1:2),target_center') - (target_radius-robot_radius),0)); */
__device__
float d_calculate_H(node d_expanded, PARAM* d_param, int mode)
{
	float heuristic = (sqrtf(2) / 3) * 
          fmaxf(0, d_param->coverage_ratio * (MAPSIZE * MAPSIZE * 4) - d_expanded.seen_map.seen_count - 4 * ROBOT_MAX);
  return heuristic;

}




//////////////////////////////////////////////////////////////////////////////////////////////////////////////////




/* param @ d_expanded - result of expanded nodes
param @ d_open - input of nodes to expand
param @ d_param - parameters about the swarm
param @ dir - Number of directions (7 for now)
param @ iteration - used for backtracing */
__global__
void k_expandStates(node* d_expanded, node* d_open, PARAM* d_param, int dir, int iteration, int real_copies, int queue_i)
{
	int dir_index = threadIdx.x;
	int robot_index = threadIdx.y;
	int node_index = blockIdx.x;
	int index = node_index * dir + dir_index;

	/* Check thread boundary */
	if (dir_index >= dir)  return;
	if (robot_index >= d_param->N) return;
	if (node_index >= real_copies) return;

	/* Expand the nodes */
	float dt = d_param->dt;
	//float ti = d_param->ti;
	float dT = d_param->dT;
	//float tf = d_param->tf;
 
  if(d_open[node_index].sequence_numel >= d_param->time_array_count) {
    d_expanded[index].isEmpty = 1;
    return;
  }

  updateSeenMap(d_open, robot_index);

	int steps = (int)dT / dt;
	int currSequenceIndex = d_open[node_index].sequence_numel;

	if(currSequenceIndex == 0) steps = 
		(int) (d_param->time_array[currSequenceIndex] / dt);
	else steps = (int) ((d_param->time_array[currSequenceIndex] - d_param->time_array[currSequenceIndex - 1]) / dt);
	/* Only write to global memory once */
	if (robot_index == 0) d_expanded[index] = d_open[node_index];

	__syncthreads();

	for (int i = 0; i < steps; i++)
	{
		node d_expanded_old = d_expanded[index];
		func[dir_index](&d_expanded[index], &d_expanded_old, robot_index, dt);
		__syncthreads();
		/* Check swarm map boundary and obstacle collision to separate valid vs invalid */
		if (d_valid_poses(d_expanded[index], d_param) == 0) {
			d_expanded[index].isEmpty = 1;
			return;
		}
		d_expanded[index].G = d_calculate_G(d_expanded[index], d_expanded_old, d_param, queue_i);
	}

	if (robot_index == 0)
	{
		/* Only write to global memory once */
		d_expanded[index].behaviorIndices[currSequenceIndex] = dir_index;
		d_expanded[index].behaviorIdx = d_expanded[index].behaviorIdx * 10 + dir_index + 1;

		/* Check swarm if reached destination */
		//d_expanded[index].reached_destination = d_target_reached(d_expanded[index], d_param);

		/* cost_estimate is total cost: cost_estimate = cost_next + H*cost_to_go */
		d_expanded[index].F = d_expanded[index].G + (d_param->H) * d_calculate_H(d_expanded[index], d_param, queue_i);
		d_expanded[index].sequence_numel += 1;
		int denom = MAPSIZE * MAPSIZE * 4;
		d_expanded[index].coverage_ratio = (float) d_expanded[index].seen_map.seen_count / denom;
		if(d_expanded[index].coverage_ratio >= d_param->coverage_ratio) d_expanded[index].reached_destination = 1;
		//printf("INSIDE KERNEL>>> [%d].F = %f, .behaviorIdx = %llu\n", 
    //        index, d_expanded[index].F, d_expanded[index].behaviorIdx);
	}
	return;
}


/////////////////////////////////////////////////////////////////////////////////////////////////////

/* HOST FUNCTIONS */

/* calculate the heuristic between curr node and the destination */
float h_calculate_H1(node d_expanded, PARAM* param, int N)
{
	float heuristic = (sqrtf(2) / 3) * 
          fmaxf(0, param->coverage_ratio * (MAPSIZE * MAPSIZE * 4) - d_expanded.seen_map.seen_count - 4 * ROBOT_MAX);
  return heuristic;
}



////////////////////////////////////////////////////////////////////////////////////////////

/* this function serves to convert position to grid index */
int convert_position(float mapsize, float robot_pos_x, float robot_pos_y)
{
	int col_offset = (int)(robot_pos_x + mapsize);
	int row_offset = (int)(mapsize - robot_pos_y);
	return row_offset * 2 * mapsize + col_offset;
}


std::vector<float> convert_index(float mapsize, int index)
{
	int row = index / (2 * mapsize);
	int col = index % (int)(2 * mapsize);
	std::vector<float> pos;
	pos.push_back(col - mapsize);
	pos.push_back(mapsize - row);

	return pos;
}


void print_distance_left(node best_node, PARAM* param)
{
	float average_x = 0;
	float average_y = 0;
	for (int i = 0; i < param->N; i++)
	{
		average_x += best_node.robot_pos[i][0];
		average_y += best_node.robot_pos[i][1];
	}
	average_x /= param->N;
	average_y /= param->N;
	float distance_left = sqrt(pow(param->target_center[0] - average_x, 2) + pow(param->target_center[1] - average_y, 2));
	printf("Distance left is %f\n", distance_left);
	return;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////


Queue* initializeQps(PARAM* param, node h_start)
{
	Queue* qps = new Queue[QUEUE_SIZE];

	for (int i = 0; i < param->q_count; i++)
	{
		qps[i].iteration = 1;
	}



	for (int i = 0; i < param->q_count; i++)
	{
    qps[i].h_open.push_back(h_start);
	}

	return qps;
}


float h_calculate_H(node d_expanded, PARAM* d_param, int mode)
{

	float heuristic = (sqrtf(2) / 3) * 
          fmaxf(0, d_param->coverage_ratio * (MAPSIZE * MAPSIZE * 4) - d_expanded.seen_map.seen_count - 4 * ROBOT_MAX);
  return heuristic;

}

void updateQueues(Queue* qps, node curr, PARAM* param, int queue_i)
{
	curr.F = curr.G + param->H * h_calculate_H(curr, param, 0);
	float queue_0_F = curr.F;
  //printf("UpdateQueues>>>> curr.F = %f, curr.behaviorIdx = %llu\n", curr.F, curr.behaviorIdx);

	qps[0].h_open.push_back(curr);
  int index = qps[0].h_open.size() - 1;
  //printf("Just to make sure UpdateQueus>>> curr.F = %f, curr.behaviorIdx = %llu\n", 
  //       qps[0].h_open[index].F, qps[0].h_open[index].behaviorIdx);
	return;
}


void expandStates(Queue* qps, PARAM* param, node* best_node_p, node* best_attempt, int queue_i)
{
	PARAM* d_param; /* device parameters */
	gpuErrchk( hipMalloc(&d_param, sizeof(PARAM)) );
	gpuErrchk( hipMemcpy(d_param, param, sizeof(PARAM), hipMemcpyHostToDevice) );

	node *d_open; /* device open nodes */
	node *d_expanded; /* device expanded nodes */

					  /* Dequeue from h_open */
	int iteration = qps[queue_i].iteration;
	//printf("Inside ExpandStates....iteration is %d\n", iteration);

	int h_open_size = qps[queue_i].h_open.size();
	int real_copies = (h_open_size > ARRAY_SIZE) ? ARRAY_SIZE : h_open_size;
	node* h_open_array = new node[real_copies]; /* temporary data used for hipMemcpy of h_open */
	node* h_expanded = new node[real_copies * DIR]; /* data used for retrieving expanded nodes data; Also used for backtracking */

	//printf("For %d, real_copies %d\n", queue_i, real_copies);

	std::copy(qps[queue_i].h_open.begin(), qps[queue_i].h_open.begin() + real_copies, h_open_array);
	/* Erase the same nodes from all queues */
	qps[queue_i].h_open.erase(qps[queue_i].h_open.begin(), qps[queue_i].h_open.begin() + real_copies);
	/* TODO: Make sure this portion of code is correct!!!! */

  //printf("Copying is done\n");
//////////////
	for (int queue_j = 0; queue_j < param->q_count; queue_j++)
	{
		if (queue_i == queue_j) continue;
	  //printf("queue_j h_open size is %d\n", qps[queue_j].h_open.size());
   	std::vector<int> erase_indices;
		for (int i = 0; i < real_copies; i++) {
			for (int j = 0; j < qps[queue_j].h_open.size(); j++)
			{
				if (h_open_array[i].behaviorIdx == qps[queue_j].h_open[j].behaviorIdx) {
          		
          			erase_indices.push_back(j);
					break;
				}
			}
		}

    	std::sort(erase_indices.begin(), erase_indices.end());
    	for (int k = erase_indices.size() - 1; k >= 0; k--)
		{
			qps[queue_j].h_open.erase(qps[queue_j].h_open.begin() + erase_indices[k]);
		}
		
	}
  
	/* Copy necessary data to device memory */
	gpuErrchk( hipMalloc(&d_open, sizeof(node) * real_copies) );
	gpuErrchk( hipMemcpy(d_open, h_open_array, sizeof(node) * real_copies, hipMemcpyHostToDevice) );
	gpuErrchk( hipMalloc(&d_expanded, sizeof(node) * real_copies * DIR) );


	/* Allocate proper blocks and threads */
	const dim3 blockSize(DIR, param->N, 1);
	const dim3 gridSize(real_copies, 1, 1);
	//const dim3 blockSize(real_copies, 1, 1);
	//const dim3 gridSize(dir, param->N, 1);

	/* Run the GPU code */
  	k_expandStates << < gridSize, blockSize >> >(d_expanded, d_open, d_param, DIR, iteration, real_copies, queue_i);
  gpuErrchk( hipPeekAtLastError() );

  	/* Copy back from GPU to CPU */
	gpuErrchk( hipMemcpy(h_expanded, d_expanded, sizeof(node) * real_copies * DIR, hipMemcpyDeviceToHost) );

	/* Update open list with expanded nodes and update best_node */
	for (int ind = 0; ind < real_copies * DIR; ind++)
	{ 
	    /* Debuggin */
	   if (h_expanded[ind].isEmpty == 0 && h_expanded[ind].reached_destination == 0)
		 {
	      //printf("After Kernel>>>>> [%d].F = %f, .behaviorIdx = %llu\n", 
	      //      ind, h_expanded[ind].F, h_expanded[ind].behaviorIdx);
			
		
			node curr = h_expanded[ind];
			float distance_to_goal = curr.F - curr.G;
	  		if(distance_to_goal < best_attempt->F - best_attempt->G) memcpy(best_attempt, &curr, sizeof(node));
				updateQueues(qps, curr, param, queue_i);
			}
			else if (h_expanded[ind].isEmpty == 0 && h_expanded[ind].reached_destination == 1
				&& h_expanded[ind].G < best_node_p->G) {
				memcpy(best_node_p, &h_expanded[ind], sizeof(node));
				//printf("UPDATING BEST NODE!! cost is %f, sequence_numel is %d\n", best_node_p->G, best_node_p->sequence_numel);
				//printf("printing sequence...\n");
				for (int i = 0; i < best_node_p->sequence_numel; i++)
				{
					int index = best_node_p->behaviorIndices[i];
					//printf("hello world????\n");
				//	printf("%s, ", behavior_array[index]);
					//printf("%d ", index);
				}
				//printf("\n");
			}
	}


	/* Sort the open_list */
	for (int i = 0; i < QUEUE_SIZE; i++)
	{
		std::sort(qps[i].h_open.begin(), qps[i].h_open.end(), [](node left, node right) {return left.F < right.F; });
	}

	/* Prepare for next iteration */
	qps[queue_i].iteration++;
	delete(h_open_array);
	hipFree(d_expanded);
	hipFree(d_open);
	hipFree(d_param);

}

__global__
void k_noSMHA(POS* d_poses, node* d_result, PARAM* d_param, int* d_sequence_end_indices)
{
	int robot_index = threadIdx.x;
	/* Error checking */
	if (robot_index >= d_param->N) return;
	
	if (robot_index == 0) {
		memcpy(d_poses[0].robot_pos, d_param->robot_pos, sizeof(float)*d_param->N * 3);
		printf("Inside k_noSMHA!!!!\n");
		printf("SEQUENCE NUMEL IS %d\n", d_result->sequence_numel);
	}
	__syncthreads();
	
	/* Expand the nodes */
	float dt = d_param->dt;
	float ti = d_param->ti;
	float dT = d_param->dT;
	//float tf = d_param->tf;
	int steps = (int)dT / dt;
	float dstart = ti;
	float dend = 0;

	int sequence_count = d_result->sequence_numel;

	updateSeenMap(d_result, robot_index);
	node d_local;
	memcpy(&d_local, d_result, sizeof(node));
	int d_poses_index = 0;

	for (int i = 0; i < sequence_count; i++)
	{
		dstart = ti + dstart + dend;
		dend = d_param->time_array[i];
		if(i == 0) steps = (int) (d_param->time_array[i] / dt);
		else steps = (int) ( (d_param->time_array[i] - d_param->time_array[i-1]) / dt);
		for (int j = 1; j <= steps; j++)
		{
			/* Forward kinematics -> Save it to POS* -> Update d_local */
			func[d_result->behaviorIndices[i]](d_result, &d_local, robot_index, dt);
			__syncthreads();
		
			if(d_valid_poses(*d_result, d_param) == 0) { d_result->isEmpty = 1;}
			d_result->G = d_calculate_G(*d_result, d_local, d_param, 0);
			if (robot_index == 0) {
				memcpy(d_poses[d_poses_index].robot_pos, d_result->robot_pos, 
				sizeof(float) * d_param->N * 3);
			}
			__syncthreads();
			memcpy(&d_local, d_result, sizeof(node));
			d_poses_index++;
		}
		d_sequence_end_indices[i] = d_poses_index - 1;
	}

  //d_result->reached_destination = d_target_reached(*d_result, d_param);
  d_result->F = d_result->G + d_param->H * d_calculate_H(*d_result, d_param, 0);
  int denom = MAPSIZE * MAPSIZE * 4;
  d_result->coverage_ratio = (float) d_result->seen_map.seen_count / denom;
  if(d_result->coverage_ratio >= d_param->coverage_ratio) d_result->reached_destination = 1;
  if(robot_index == 0)
  {
  	printf("\n");
  	for(int i = 0 ; i < MAPSIZE * 2; i++)
  	{
  		for(int j = 0; j < MAPSIZE * 2; j++)
  		{
  			if(d_result->seen_map.seen[i][j] == 0)
  			{
  				printf("*");
  			}
  			else{
  				printf("+");
  			}
  		}
  		printf("\n");
  	}
  	printf("seen_count %d, denom %d\n", d_result->seen_map.seen_count, denom);
  	printf("coverage_ratio is %f\n", d_result->coverage_ratio);
  }

  

	return;
}


void noSMHAstar(PARAM* param, RETURN* return_1, node* result_node)
{
	/* Only for debugging purposes need to make sure that all the info is delivered correctly */
	printf("Inside noSMHAstar.....\n");
	printf("number of switch times given.... %d\n", param->time_array_count);
	for(int i = 0; i < param->time_array_count; i++)
	{
		printf("%f ", param->time_array[i]);
	}
	printf("\n");

	printf("checking result node also.... sequence_numel is %d\n", result_node->sequence_numel);
	for(int i = 0; i < param->sequence_array_count; i++)
	{
		printf("%s %d", behavior_array[result_node->behaviorIndices[i]], result_node->behaviorIndices[i]);
	}
	printf("\n");



	PARAM* d_param; /* device parameters */
	gpuErrchk(hipMalloc(&d_param, sizeof(PARAM)));
	gpuErrchk(hipMemcpy(d_param, param, sizeof(PARAM), hipMemcpyHostToDevice));

	node* d_result;
	gpuErrchk(hipMalloc(&d_result, sizeof(node)));
	gpuErrchk(hipMemcpy(d_result, result_node, sizeof(node), hipMemcpyHostToDevice));


	int seq_n = result_node->sequence_numel;
    printf("param->tf = %f, param->dt = %f\n", param->tf, param->dt);
	int h_result_size = param->tf / param->dt + 1;
	printf("h_result_size is ... %d\n", h_result_size);
	POS* h_poses = new POS[h_result_size];
	POS* d_poses;
	gpuErrchk(hipMalloc(&d_poses, sizeof(POS) * h_result_size));

	
	/* Allocate space on device for sequence_end_indices array, cost_of_path, is_valid_path */
	//float cost_of_path = 0;
	//int is_valid_path = 0;
	int* h_sequence_end_indices = new int[param->sequence_array_count];
	int* d_sequence_end_indices;
	gpuErrchk(hipMalloc(&d_sequence_end_indices, sizeof(int) * (param->sequence_array_count)));



	const dim3 gridSize(1, 1, 1);
	const dim3 blockSize(param->N, 1, 1);
	k_noSMHA << <gridSize, blockSize >> >(d_poses, d_result, d_param, d_sequence_end_indices);
    gpuErrchk( hipPeekAtLastError() );

	/* Copy back from GPU to CPU */
	hipMemcpy(h_poses, d_poses, sizeof(POS) * h_result_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_sequence_end_indices, d_sequence_end_indices, 
		   sizeof(int) * (param->sequence_array_count), hipMemcpyDeviceToHost);


    node* h_result = new node[1];
	hipMemcpy(h_result, d_result, sizeof(node), hipMemcpyDeviceToHost);

	
	printf("returned from kernel...\n");
	printf("printing all necessary information to check that kernel returned correctly\n");
	printf("Cost of path(G) = %f\n", h_result->G);
    printf("h_result->isEmpty %d\n", h_result->isEmpty);
	if(h_result->isEmpty == 0) printf("PATH IS VALID\n");
 	else printf("PATH IS INVALID\n");

	/* Copy all the necessary information back to return struct */
  return_1->cost_of_path = h_result->G;
	if(h_result->isEmpty) return_1->is_valid_path = 0;
	else return_1->is_valid_path = 1;
	return_1->is_complete = h_result->reached_destination;
	return_1->is_optimal = h_result->optimal;
	return_1->coverage_ratio = h_result->coverage_ratio;
	return_1->seen_map = h_result->seen_map;

  printf("updating robot_positions...\n");
  printf("h_result_size is %d\n", h_result_size);
	for(int i = 0; i < h_result_size; i++)
	{
    //printf("[%d] x,y,r = (%f %f %f)\n",i, h_poses[i].robot_pos[0][0], h_poses[i].robot_pos[0][1],
				//	h_poses[i].robot_pos[0][2]); 
		return_1->robot_positions.push_back(h_poses[i]);
	}
  printf("updating sequence end indices and sequence string array\n");
  printf("seq_n is %d\n", seq_n);
	for(int i = 0; i < seq_n; i++)
	{
		return_1->sequence_end_indices.push_back(h_sequence_end_indices[i]);
		return_1->sequence_string_array.push_back(behavior_array_display[h_result->behaviorIndices[i]]);
	}

	printf("making sure that return struct has necessary info...\n");
	printf("return_1->cost_of_path = %f\n", return_1->cost_of_path);
	printf("return-1->is_valid_path = %d\n", (int) return_1->is_valid_path);
	printf("return_1->coverage_ratio = %f\n", return_1->coverage_ratio);
  for(int i = 0; i < seq_n; i++)
  {
    printf("return_1->sequence_end_indices[%d] = %d\n", 
            i, return_1->sequence_end_indices[i]);
  }

  memcpy(result_node, h_result, sizeof(node));
  delete h_result;
	return;
}


/* returns @ string of behavior sequence (for now) behavior__behavior__behavior__....behavior*/
node SMHAstar(PARAM* param, node h_start)
{
	clock_t start = clock();
	cout << "Starting IMHA star " << endl;

	Queue* qps = initializeQps(param, h_start);
	node result;
	result.isEmpty = 1;
	result.F = numeric_limits<float>::max();

	node* best_node_p = new node[1];
	best_node_p->reached_destination = 0;
	best_node_p->isEmpty = 1;
	best_node_p->F = numeric_limits<float>::max();
	best_node_p->G = numeric_limits<float>::max();
	best_node_p->N = param->N;

	node best_attempt; //criteria is distance to destination
	memcpy(&best_attempt, &h_start, sizeof(node));

	//convert double to float for time_arra
	while (!qps[0].h_open.empty())
	{
		/* Exit if exceeded the amount of time */
		clock_t end = clock();
		float time_elapsed = float(end - start);

		if (time_elapsed > 10000000) //50 sec
		{
			printf("Exceeded time limit of %f (ms)", time_elapsed);
			if(result.isEmpty && best_node_p->isEmpty) 
			memcpy(&result, &best_attempt, sizeof(node));
			else if(result.isEmpty)
			memcpy(&result, best_node_p, sizeof(node));

			return result;
		}
  		
		
        node* minKeys = &qps[0].h_open[0];
				

		
		int size = qps[0].h_open.size() > ARRAY_SIZE ? ARRAY_SIZE : qps[0].h_open.size();
		for(int i = 0; i < size; i++)
		{
			node minKey_0 = minKeys[i];
			if (best_node_p->F <= minKey_0.F && minKey_0.coverage_ratio >= param->coverage_ratio)
			{
				printf("DONE WITH SEARCH!! NOW PRINTING RESULTS\n");
				printf("Current best node cost %f, sequence_numel is %d\n", best_node_p->G, best_node_p->sequence_numel);
				print_distance_left(*best_node_p, param);

				printf("printing sequence...\n");
				for (int i = 0; i < best_node_p->sequence_numel; i++)
				{
					int index = best_node_p->behaviorIndices[i];
					printf("%s, ", behavior_array[index]);
	 			}
				printf("\n");

				memcpy(&result, best_node_p, sizeof(node));
				clock_t end = clock();
				float time_elapsed = float(end - start);
				cout << "Time to calculate the route (ms): " << time_elapsed << endl;
				printf("inside smha!!! result sequence_numel!!!%d\n", result.sequence_numel);
				result.optimal = true;
				return result;
				
			}
		}

		expandStates(qps, param, best_node_p, &best_attempt, 0);
		
			

	}


	if (best_node_p->isEmpty == 0 && best_node_p->G < result.G) memcpy(&result, best_node_p, sizeof(node));
	

  if (result.isEmpty == 1 && best_node_p->isEmpty == 0) {
    memcpy(&result, best_node_p, sizeof(node));
    result.optimal = true;
  }

	if (result.isEmpty == 1 && best_node_p->isEmpty == 1) {
		printf("no route found, returning the best attempt\n");
		memcpy(&result, &best_attempt, sizeof(node));
	}

	else {
		printf("printing sequence...\n");
		for (int i = 0; i < result.sequence_numel; i++)
		{
			int index = result.behaviorIndices[i];
			printf("%s, ", behavior_array[index]);
		}
		printf("\n");
	}


	clock_t end = clock();
	float time_elapsed = float(end - start);
	cout << "Time to calculate the route (ms): " << time_elapsed << endl;
	return result;


}


__global__
void k_SAVE(POS* d_poses, node* d_result, node* d_start, PARAM* d_param)
{
	int robot_index = threadIdx.x;
	/* Error checking */
	if (robot_index >= d_param->N) return;

	if (robot_index == 0) memcpy(d_poses[0].robot_pos, d_param->robot_pos, sizeof(float)*d_param->N * 3);

	__syncthreads();

	/* Expand the nodes */
	float dt = d_param->dt;
	//float ti = d_param->ti;
	float dT = d_param->dT;
	//float tf = d_param->tf;
	int steps = (int)dT / dt;

	int sequence_count = d_result->sequence_numel;

	node d_local;
	memcpy(&d_local, d_start, sizeof(node));
	for (int i = 0; i < sequence_count; i++)
	{

		for (int j = 1; j <= steps; j++)
		{
			/* Forward kinematics -> Save it to POS* -> Update d_local */
			func[d_result->behaviorIndices[i]](d_start, &d_local, robot_index, dt);
			__syncthreads();
			if (robot_index == 0) memcpy(d_poses[i*steps + j].robot_pos, d_start->robot_pos, sizeof(float) * d_param->N * 3);
			__syncthreads();
			memcpy(&d_local, d_start, sizeof(node));
		}
	}


	return;

}

void SAVE_launch(node result_node, RETURN* return_1, PARAM* param)
{
	/* Open file and retrieve file id */

	printf("Printing out the sequence...\n");
	for(int i = 0; i < result_node.sequence_numel; i++)
	{
		printf("%s ", behavior_array[result_node.behaviorIndices[i]]);
	}
	printf("\n");

	PARAM* d_param; /* device parameters */
	gpuErrchk(hipMalloc(&d_param, sizeof(PARAM)));
	gpuErrchk(hipMemcpy(d_param, param, sizeof(PARAM), hipMemcpyHostToDevice));

	node* d_result;
	gpuErrchk(hipMalloc(&d_result, sizeof(node)));
	gpuErrchk(hipMemcpy(d_result, &result_node, sizeof(node), hipMemcpyHostToDevice));

	node h_start;
	h_start.isEmpty = 0;
	h_start.N = param->N;
	h_start.sequence_numel = 0;
	memcpy(&h_start.robot_pos, &param->robot_pos, sizeof(float)*ROBOT_MAX * 3);


	/* DEBUGGING!!!! */
	printf("making sure h_start has correct positions.....\n");
	for (int i = 0; i < param->N; i++)
	{
		printf("(%f %f %f) ", h_start.robot_pos[i][0], h_start.robot_pos[i][1], h_start.robot_pos[i][2]);
	}
	printf("\n");
	/* End of debugging.... */

	node* d_start;
	gpuErrchk(hipMalloc(&d_start, sizeof(node)));
	gpuErrchk(hipMemcpy(d_start, &h_start, sizeof(node), hipMemcpyHostToDevice));


	int steps = (int)param->dT / param->dt;
	int seq_n = result_node.sequence_numel;
	int h_result_size = steps* seq_n + 1;
	printf("steps, seq_n, %d, %d\n", steps, seq_n);
	printf("h_result_size is ... %d\n", h_result_size);
	POS* h_poses = new POS[h_result_size];
	POS* d_poses;
	gpuErrchk(hipMalloc(&d_poses, sizeof(POS) * h_result_size));




	const dim3 gridSize(1, 1, 1);
	const dim3 blockSize(param->N, 1, 1);
	k_SAVE << <gridSize, blockSize >> >(d_poses, d_result, d_start, d_param);
  gpuErrchk( hipPeekAtLastError() );

	/* Copy back from GPU to CPU */
	hipMemcpy(h_poses, d_poses, sizeof(POS) * h_result_size, hipMemcpyDeviceToHost);

	/* Now save everything onto the txt */
	cout << "saving everything in txt file" << endl;


	//output_f = fopen(output_fname.str().c_str(), "w");
	int times = 0;
	printf("H_RESULT_SIZE IS %d\n", h_result_size);
	for (int i = 0; i < h_result_size; i++)
	{
		for (int j = 0; j < param->N; j++)
		{
			//fprintf(output_f, (std::to_string(h_poses[i].robot_pos[j][0]) + " " + std::to_string(h_poses[i].robot_pos[j][1])
			//	+ " " + std::to_string(h_poses[i].robot_pos[j][2]) + "\n").c_str());
			times++;
		}
		//fprintf(output_f, "\n");
	}
	return;
}

void SMHAstar_wrapper(PARAM* param, RETURN* result_1)
{
  printf("Inside smhastar_wrapper\n");
	node h_start; //starting node

	h_start.isEmpty = 0;
	h_start.N = param->N;
	h_start.sequence_numel = param->sequence_array_count;
  h_start.behaviorIdx = 0;
  for(int i = 0; i < h_start.sequence_numel; i++)
  {
    h_start.behaviorIndices[i] = param->sequence_array[i];
  }

	printf("Inside smhastar_wrapper, param->N = %d\n", param->N);

	std::copy(&param->robot_pos[0][0], &param->robot_pos[0][0] + param->N * 3, &h_start.robot_pos[0][0]);

	printf("Done copying robot_positions param->h_start\n");
	h_start.F = param->H * h_calculate_H1(h_start, param, h_start.N);
	h_start.G = 0;
	h_start.reached_destination = 0;

	h_start.seen_map.seen_count = 0;
	h_start.seen_map.mapsize = MAPSIZE;
	for(int i = 0; i < MAPSIZE*2; i++)
	{
		for(int j = 0; j < MAPSIZE*2; j++)
		{
			h_start.seen_map.seen[i][j] = 0;
		}
	}
	

  RETURN result_tmp;
  if(h_start.sequence_numel != 0){

    noSMHAstar(param, result_1, &h_start);
    if(param->time_array_count == param->sequence_array_count) return;
  }
////////////////////////////////////////////////////////////////////

  //Need to check that this works okay...
  printf("Printing h_start information insdie smha_star_wrapper\n");
  printf("isEmpty = %d, N = %d, F = %f, G = %f, behaviorIdx = %llu, reached_destination = %d, sequence_numel = %d\n", h_start.isEmpty,
          h_start.N, h_start.F, h_start.G, h_start.behaviorIdx,
          h_start.reached_destination, h_start.sequence_numel);
  for(int i = 0; i < h_start.sequence_numel; i++)
  {
    printf("%s ", behavior_array[h_start.behaviorIndices[i]]);
  }
  printf("\n");
  for(int i = 0; i < h_start.N; i++)
  {
    printf("robot[%d] x,y,r = %f %f %f\n", i, h_start.robot_pos[i][0],
			h_start.robot_pos[i][1], h_start.robot_pos[i][2]);
  }

  if(h_start.isEmpty == 1)
  { /*
    printf("H_start is empty!!!! making sure result_tmp has all the correct information....\n");
    for(int i = 0; i < result_tmp.sequence_end_indices.size(); i++)
    {
      printf("sequence[%d] end index is %d\n", i, result_tmp.sequence_end_indices[i]);
    }

    
    memcpy(result_1, &result_tmp, sizeof(RETURN));
		for(int i = 0; i < result_1->sequence_end_indices.size(); i++)
    {
      printf("sequence[%d] end index is %d\n", i, result_1->sequence_end_indices[i]);
    }
    */
    printf("RETURNING EARLY!!!\n");
    return;
  }  
  

	node result_node = SMHAstar(param, h_start); //note this result_node might be simply the closest attempt to the goal

  printf("\n\n\n\n\n\n\n");
  printf("After returning from SMHAstar...\n"); 
  printf("isEmpty = %d, N = %d, F = %f, G = %f, behaviorIdx = %llu, reached_destination = %d, sequence_numel = %d\n", result_node.isEmpty,
          result_node.N, result_node.F, result_node.G, result_node.behaviorIdx,
          result_node.reached_destination, result_node.sequence_numel);
  for(int i = 0; i < result_node.sequence_numel; i++)
  {
    printf("%s ", behavior_array[result_node.behaviorIndices[i]]);
  }
  printf("\n");
  for(int i = 0; i < result_node.N; i++)
  {
    printf("robot[%d] x,y,r = %f %f %f\n", i, result_node.robot_pos[i][0],
			result_node.robot_pos[i][1], result_node.robot_pos[i][2]);
  }
//////////////////////////////////////////////////////////////

	node h_final; //starting node

	h_final.isEmpty = 0;
	h_final.N = param->N;
	h_final.sequence_numel = result_node.sequence_numel;
  h_start.behaviorIdx = 0;
  for(int i = 0; i < h_final.sequence_numel; i++)
  {
    h_final.behaviorIndices[i] = result_node.behaviorIndices[i];
  }


	std::copy(&param->robot_pos[0][0], &param->robot_pos[0][0] + param->N * 3, &h_final.robot_pos[0][0]);

	printf("Done copying robot_positions param->h_start\n");
	h_final.F = param->H * h_calculate_H1(h_final, param, h_final.N);
	h_final.G = 0;
	h_final.reached_destination = 0;

	h_final.seen_map.seen_count = 0;
	h_final.seen_map.mapsize = MAPSIZE;
	for(int i = 0; i < MAPSIZE*2; i++)
	{
		for(int j = 0; j < MAPSIZE*2; j++)
		{
			h_final.seen_map.seen[i][j] = 0;
		}
	}	

	printf("checking the sequence before entering noSMHAstar...\n");
	printf("h_final sequence count is %d\n", h_final.sequence_numel);
	for(int i = 0; i < h_final.sequence_numel; i++)
	{
		printf("%s ", behavior_array_display[h_final.behaviorIndices[i]].c_str());

	}
	printf("\n");

  param->sequence_array_count = h_final.sequence_numel;
  for(int i = 0; i < h_final.sequence_numel; i++)
  {
    param->sequence_array[i] = h_final.behaviorIndices[i];
  }

  result_1->robot_positions.clear();
  result_1->sequence_end_indices.clear();
  result_1->sequence_string_array.clear();

	noSMHAstar(param, result_1, &h_final);
	return;



}




void onlySMHAstar_wrapper(PARAM* param, RETURN* result_1)
{
  printf("Inside smhastar_wrapper\n");
	node h_start; //starting node

	h_start.isEmpty = 0;
	h_start.N = param->N;
	h_start.sequence_numel = param->sequence_array_count;
	h_start.behaviorIdx = 0;
  for(int i = 0; i < h_start.sequence_numel; i++)
  {
    h_start.behaviorIndices[i] = param->sequence_array[i];
  }

	printf("Inside smhastar_wrapper, param->N = %d\n", param->N);

	std::copy(&param->robot_pos[0][0], &param->robot_pos[0][0] + param->N * 3, &h_start.robot_pos[0][0]);

	printf("Done copying robot_positions param->h_start\n");
	h_start.F = param->H * h_calculate_H1(h_start, param, h_start.N);
	h_start.G = 0;
	h_start.reached_destination = 0;

	h_start.seen_map.seen_count = 0;
	h_start.seen_map.mapsize = MAPSIZE;
	for(int i = 0; i < MAPSIZE*2; i++)
	{
		for(int j = 0; j < MAPSIZE*2; j++)
		{
			h_start.seen_map.seen[i][j] = 0;
		}
	}
	

	node result_node = SMHAstar(param, h_start); //note this result_node might be simply the closest attempt to the goal

  printf("\n\n\n\n\n\n\n");
  printf("After returning from SMHAstar...\n"); 
  printf("isEmpty = %d, N = %d, F = %f, G = %f, behaviorIdx = %llu, reached_destination = %d, sequence_numel = %d\n", result_node.isEmpty,
          result_node.N, result_node.F, result_node.G, result_node.behaviorIdx,
          result_node.reached_destination, result_node.sequence_numel);
  for(int i = 0; i < result_node.sequence_numel; i++)
  {
    printf("%s ", behavior_array[result_node.behaviorIndices[i]]);
  }
  printf("\n");
  for(int i = 0; i < result_node.N; i++)
  {
    printf("robot[%d] x,y,r = %f %f %f\n", i, result_node.robot_pos[i][0],
			result_node.robot_pos[i][1], result_node.robot_pos[i][2]);
  }
//////////////////////////////////////////////////////////////

	node h_final; //starting node

	h_final.isEmpty = 0;
	h_final.N = param->N;
	h_final.sequence_numel = result_node.sequence_numel;
  h_start.behaviorIdx = 0;
  for(int i = 0; i < h_final.sequence_numel; i++)
  {
    h_final.behaviorIndices[i] = result_node.behaviorIndices[i];
  }


	std::copy(&param->robot_pos[0][0], &param->robot_pos[0][0] + param->N * 3, &h_final.robot_pos[0][0]);

	printf("Done copying robot_positions param->h_start\n");
	h_final.F = param->H * h_calculate_H1(h_final, param, h_final.N);
	h_final.G = 0;
	h_final.reached_destination = 0;

	h_final.seen_map.seen_count = 0;
	h_final.seen_map.mapsize = MAPSIZE;
	for(int i = 0; i < MAPSIZE*2; i++)
	{
		for(int j = 0; j < MAPSIZE*2; j++)
		{
			h_final.seen_map.seen[i][j] = 0;
		}
	}	

	printf("checking the sequence before entering noSMHAstar...\n");
	printf("h_final sequence count is %d\n", h_final.sequence_numel);
	for(int i = 0; i < h_final.sequence_numel; i++)
	{
		printf("%s ", behavior_array_display[h_final.behaviorIndices[i]].c_str());

	}
	printf("\n");

  param->sequence_array_count = h_final.sequence_numel;
  for(int i = 0; i < h_final.sequence_numel; i++)
  {
    param->sequence_array[i] = h_final.behaviorIndices[i];
  }

  result_1->robot_positions.clear();
  result_1->sequence_end_indices.clear();
  result_1->sequence_string_array.clear();

	noSMHAstar(param, result_1, &h_final);
	return;

}





/////////////////////////////////////////////////////////////////////////////////////////////////////////

void initialize_parameters(PARAM* param, std::vector<float> time_array, 
	std::vector<int>sequence_array, std::vector<uint8_t> fix_array, float coverage_ratio)
{ 
	param->time_array_count = time_array.size();
  param->sequence_array_count = sequence_array.size();
  param->coverage_ratio = coverage_ratio;

  printf("Inside initialize_params....\n");
	printf("Length of time_array is %d, Length of sequence is %d\n", time_array.size(), sequence_array.size());

	for(int i = 0; i < time_array.size(); i++)
	{
		param->time_array[i] = time_array[i];
	}
	for(int i = 0; i < sequence_array.size(); i++)
	{
		param->sequence_array[i] = sequence_array[i];
	}
  /*
  for(int i = 0; i < param->N; i++)
	{
		printf("robot %d pos = %f %f %f\n", i, param->robot_pos[i][0], param->robot_pos[i][1],
						param->robot_pos[i][2]);
	}

	for(int i = 0; i < param->M; i++)
	{
		printf("obstacle %d pos = %f %f %f\n", i, param->obstacle_pos[i][0], param->obstacle_pos[i][1],
					param->obstacle_pos[i][2]);
	}
  */
  printf("Returning from init params\n");
	return;
}


void fix_robot_positions(PARAM* param)
{
	param->N = 10;
	param->robot_pos[0][0] = -18;
	param->robot_pos[0][1] = -10;
	param->robot_pos[0][2] = 0;

	param->robot_pos[1][0] = -17;
	param->robot_pos[1][1] = -10;
	param->robot_pos[1][2] = 0;

	param->robot_pos[2][0] = -16;
	param->robot_pos[2][1] = -10;
	param->robot_pos[2][2] = 0;

	param->robot_pos[3][0] = -15;
	param->robot_pos[3][1] = -10;
	param->robot_pos[3][2] = 0;

	param->robot_pos[4][0] = -14;
	param->robot_pos[4][1] = -10;
	param->robot_pos[4][2] = 0;

	param->robot_pos[5][0] = -18;
	param->robot_pos[5][1] = -11;
	param->robot_pos[5][2] = 0;

	param->robot_pos[6][0] = -17;
	param->robot_pos[6][1] = -11;
	param->robot_pos[6][2] = 0;

	param->robot_pos[7][0] = -16;
	param->robot_pos[7][1] = -11;
	param->robot_pos[7][2] = 0;

	param->robot_pos[8][0] = -15;
	param->robot_pos[8][1] = -11;
	param->robot_pos[8][2] = 0;

	param->robot_pos[9][0] = -14;
	param->robot_pos[9][1] = -11;
	param->robot_pos[9][2] = 0;
}

void fix_obstacle_positions(PARAM* param)
{
	param->M = 1;
	param->obstacle_pos[0][0] = 0;
	param->obstacle_pos[0][1] = 0;
	param->obstacle_pos[0][2] = 5;

	param->obstacle_pos[1][0] = -10;
	param->obstacle_pos[1][1] = 10;
	param->obstacle_pos[1][2] = 7;
}





void initialize_result(node* result_node, PARAM* param)
{
	result_node->isEmpty = 0;
	result_node->N = param->N;
	result_node->sequence_numel = param->time_array_count;
	memcpy(&result_node->robot_pos[0][0], &param->robot_pos[0][0], sizeof(float) * param->N * 3);
	result_node->F = param->H * h_calculate_H1(*result_node, param, result_node->N);
	result_node->G = 0;
	result_node->reached_destination = 0;
	memcpy(&result_node->behaviorIndices, &param->sequence_array, sizeof(int)*SEQ_MAX);
	result_node->seen_map.seen_count = 0;
	result_node->seen_map.mapsize = MAPSIZE;
	for(int i = 0; i < MAPSIZE*2; i++)
	{
		for(int j = 0; j < MAPSIZE*2; j++)
		{
			result_node->seen_map.seen[i][j] = 0;
		}
	}
	return;
}

RETURN testmain(PARAM* param, int isAided, std::vector<float> time_array, 
	std::vector<int> sequence_array, std::vector<uint8_t> isFixed, float coverage_ratio)
{
	printf("starting\n");

	clock_t start;
    double duration;

    start = std::clock();

    RETURN return_1;

	initialize_parameters(param, time_array, sequence_array, isFixed, coverage_ratio);

	if(isAided == 1) SMHAstar_wrapper(param, &return_1);
	else if(isAided == 0){
		node result_node;
		initialize_result(&result_node, param);
	
		noSMHAstar(param, &return_1, &result_node);
		//printf("after returning from noSMHAstar function....\n");
		//printf("return_1 cost_of_path = %f\n", return_1.cost_of_path);
	}
	else if(isAided == 1)
	{
		onlySMHAstar_wrapper(param, &return_1);
	}

    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

    std::cout<<"Elapsed: "<< duration << " Seconds" << '\n';


	return return_1;
}
