#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 Sasanka Nagavalli
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include <cmath>

#include "hipcub/hipcub.hpp"

#include "common.h"

#include "utility.cuh"

#include "formation_control.cuh"

__device__
void behavior_formation_control_stage1(BehaviorContext * context, int i, int j) {
  BehaviorFormationControlContext * ctx = 
    static_cast<BehaviorFormationControlContext *>(context);
  typedef hipcub::BlockReduce<float, ROBOTS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduce;
  __shared__ typename BlockReduce::TempStorage storage;

  float dx = ctx->poses.x[j] - ctx->poses.x[i];
  float dy = ctx->poses.y[j] - ctx->poses.y[i];
  float fx = ctx->formation_x[j] - ctx->formation_x[i];
  float fy = ctx->formation_y[j] - ctx->formation_y[i];
  float distance = sqrtf(dx*dx + dy*dy);
  float delta_x = dx - fx;
  float delta_y = dy - fy;
  int c = (distance < CONNECTIVITY_RADIUS);
  float n = BlockReduce(storage).Sum(c); __syncthreads();
  float vx = BlockReduce(storage).Sum(delta_x*c); __syncthreads();
  float vy = BlockReduce(storage).Sum(delta_y*c); __syncthreads();

  if (j == 0) {
    ctx->velocity_x[i] = vx / n;
    ctx->velocity_y[i] = vy / n;
  }
}

__device__
void behavior_formation_control_stage2(BehaviorContext * context, int i) {
  BehaviorFormationControlContext * ctx = 
    static_cast<BehaviorFormationControlContext *>(context);
  float vx = ctx->velocity_x[i];
  float vy = ctx->velocity_y[i];
  float theta = ctx->poses.theta[i];
  float bx = cosf(theta);
  float by = sinf(theta);
  float w = wrapToPi(atan2f(vy, vx) - theta);
  ctx->control_v[i] = GAIN_V * (vx*bx + vy*by);
  ctx->control_w[i] = GAIN_W * w;
}

